#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"
#define MAX_PASSPHRASE_LENGTH 5 // "book" test word + null terminator. DON'T FORGET TO INCREASE
#include "p_chain.h"
#include "nlohmann/json.hpp"
#include <cstring>
#include <string.h>
#include <limits.h>
#include <nvtx3/nvToolsExt.h>

#define P_CHAIN_ADDRESS_LENGTH 45  // Assuming the p-chain address is 45 characters long

__device__ bool d_address_found = false;
__device__ char d_address_value[P_CHAIN_ADDRESS_LENGTH + 1];
__device__ char d_passphrase_value[MAX_PASSPHRASE_LENGTH];

#define OVERFLOW_FLAG ULLONG_MAX

struct ThreadTiming {
    int blockIdx;
    int threadIdx;
    long long startTime;
    long long endTime;
};

unsigned long long find_variant_id(const char* s) {
    const char* alphabet = "abcdefghijklmnopqrstuvwxyz";
    int base = strlen(alphabet);
    unsigned long long result = 0;
    unsigned long long prev_result = 0;
    
    for (int i = 0; s[i] != '\0'; i++) {
        const char* pos = strchr(alphabet, s[i]);
        if (pos != NULL) {
            int index = pos - alphabet;
            
            // Check for multiplication overflow
            if (result > ULLONG_MAX / base) {
                return OVERFLOW_FLAG;
            }
            result *= base;
            
            // Check for addition overflow
            if (result > ULLONG_MAX - (index + 1)) {
                return OVERFLOW_FLAG;
            }
            result += index + 1;
            
            // Check if the value wrapped around
            if (result < prev_result) {
                return OVERFLOW_FLAG;
            }
            
            prev_result = result;
        }
    }
    
    return result;
}

// __device__ __forceinline__ void find_letter_variant(int variant_id, char* passphrase_value) {
__device__ void find_letter_variant(int variant_id, char* passphrase_value) {
    // Define alphabet as a constant array
    const char alphabet[] = "abcdefghijklmnopqrstuvwxyz";
    const int alphabet_length = 26;

    // Initialize first character to null terminator, rest will be filled as needed
    passphrase_value[0] = '\0';

    // Handle the special case for variant_id == 0
    if (variant_id == 0) {
        passphrase_value[0] = alphabet[0];
        passphrase_value[1] = '\0';
        return;
    }

    int result_length = 0;
    
    // Generate the passphrase
    while (variant_id > 0 && result_length < MAX_PASSPHRASE_LENGTH - 1) {  // Leave room for null terminator
        --variant_id;  // Adjust for 0-based indexing
        passphrase_value[result_length++] = alphabet[variant_id % alphabet_length];
        variant_id /= alphabet_length;
    }
    passphrase_value[result_length] = '\0';  // Ensure null termination

    // Reverse the result in-place
    int start = 0;
    int end = result_length - 1;
    while (start < end) {
        char temp = passphrase_value[start];
        passphrase_value[start] = passphrase_value[end];
        passphrase_value[end] = temp;
        ++start;
        --end;
    }
}

__device__ int my_strncmp(const char* s1, const char* s2, size_t n) {
    for (size_t i = 0; i < n; ++i) {
        if (s1[i] != s2[i]) {
            return s1[i] - s2[i];
        }
        if (s1[i] == '\0') {
            return 0;
        }
    }
    return 0;
}

__global__ void variant_kernel(
    unsigned long long start_variant_id,
    unsigned long long end_variant_id, 
    const char *expected_value, 
    const char *mnemonic,
    ThreadTiming *timings
) {
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;
    int globalIdx = blockId * blockDim.x + threadId;
    unsigned long long variant_id = start_variant_id + globalIdx;
    
    // Record start time
    long long start_time = clock64();
    timings[globalIdx].blockIdx = blockId;
    timings[globalIdx].threadIdx = threadId;
    timings[globalIdx].startTime = start_time;
    
    while (variant_id <= end_variant_id && !d_address_found) {
        char local_passphrase_value[MAX_PASSPHRASE_LENGTH] = {0};
        find_letter_variant(variant_id, local_passphrase_value);
        
        // Calculate p-chain address
        P_CHAIN_ADDRESS_STRUCT p_chain_address = restore_p_chain_address((uint8_t*)mnemonic, local_passphrase_value);
        
         if (my_strncmp(p_chain_address.data, expected_value, P_CHAIN_ADDRESS_LENGTH+1) == 0) {
            d_address_found = true;
            for (int i = 0; i < P_CHAIN_ADDRESS_LENGTH; i++) {
                d_address_value[i] = p_chain_address.data[i];
            }
            // Set the passphrase value
            for (int i = 0; i < MAX_PASSPHRASE_LENGTH; i++) {
                d_passphrase_value[i] = local_passphrase_value[i];
            }
            d_address_value[P_CHAIN_ADDRESS_LENGTH] = '\0';
        }
        // Early exit if address is found
        if (d_address_found) break;
        
        variant_id += gridDim.x * blockDim.x;
    }
    
    // Record end time
    long long end_time = clock64();
    timings[globalIdx].endTime = end_time;
}

void write_timing_to_csv(const char* filename, ThreadTiming* timings, int num_threads) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error opening file: " << filename << std::endl;
        return;
    }
    
    file << "BlockIdx,ThreadIdx,StartTime,EndTime,Duration" << std::endl;
    
    for (int i = 0; i < num_threads; i++) {
        file << timings[i].blockIdx << ","
             << timings[i].threadIdx << ","
             << timings[i].startTime << ","
             << timings[i].endTime << ","
             << (timings[i].endTime - timings[i].startTime) << std::endl;
    }
    
    file.close();
    std::cout << "Timing data saved to " << filename << std::endl;
}

unsigned long long calculate_iterations(unsigned long long start_variant_id, unsigned long long end_variant_id, int h_max_threads) {
    unsigned long long search_area = end_variant_id - start_variant_id;
    return (search_area + h_max_threads - 1) / h_max_threads;
}

int main() {
    // int threadsPerBlock = 256;
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;
    int totalThreads = threadsPerBlock * blocksPerGrid;

    // Allocate memory for timing data
    ThreadTiming *h_timings = new ThreadTiming[totalThreads];
    ThreadTiming *d_timings;
    hipMalloc(&d_timings, totalThreads * sizeof(ThreadTiming));

    bool h_address_found = false;
    char h_address_value[P_CHAIN_ADDRESS_LENGTH + 1];
    char h_passphrase_value[MAX_PASSPHRASE_LENGTH];

    // Read expected value from JSON file
    std::ifstream config_file("config.json");
    if (!config_file.is_open()) {
        std::cerr << "Failed to open config.json" << std::endl;
        return -1;
    }
    
    nlohmann::json config;
    config_file >> config;
    
    // std::string expected_value = config["p_chain_address"];
    std::string expected_value = config["p_chain_address"];
    std::string mnemonic = config["mnemonic"];
    std::string start_passphrase = config["start_passphrase"];
    std::string end_passphrase = config["end_passphrase"];

    if (expected_value.length() != P_CHAIN_ADDRESS_LENGTH) {
        std::cerr << "Invalid p_chain_address length in config.json" << std::endl;
        return -1;
    }
    if (mnemonic.empty()) {
        std::cerr << "Mnemonic is empty in config.json" << std::endl;
        return -1;
    }

    // Calculate search area
    unsigned long long start_variant_id = find_variant_id(start_passphrase.c_str());
    unsigned long long end_variant_id = find_variant_id(end_passphrase.c_str());

    if (start_variant_id == OVERFLOW_FLAG || end_variant_id == OVERFLOW_FLAG) {
        std::cerr << "Passphrase overflow detected. The maximum passphrase is gkgwbylwrxtlpn" << std::endl;
        return -1;
    }

    std::cout << "Start variant id: " << start_variant_id << std::endl;
    std::cout << "End variant id: " << end_variant_id << std::endl;
    std::cout << "Search area: " << end_variant_id - start_variant_id + 1 << std::endl;

    char *d_expected_value;
    hipMalloc((void**)&d_expected_value, P_CHAIN_ADDRESS_LENGTH + 1);
    hipMemcpy(d_expected_value, expected_value.c_str(), P_CHAIN_ADDRESS_LENGTH + 1, hipMemcpyHostToDevice);

    char *d_mnemonic;
    hipMalloc((void**)&d_mnemonic, mnemonic.length() + 1);
    hipMemcpy(d_mnemonic, mnemonic.c_str(), mnemonic.length() + 1, hipMemcpyHostToDevice);

    std::cout << "Launching kernel with " << blocksPerGrid << " blocks and " << threadsPerBlock << " threads per block" << std::endl;
    
    // Start NVTX range
    nvtxRangePush("KernelExecution");

    // Launch kernel
    variant_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        start_variant_id, 
        end_variant_id, 
        d_expected_value, 
        d_mnemonic,
        d_timings
    );

    // End NVTX range
    nvtxRangePop();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error launching kernel: %s\n", hipGetErrorString(err));
        hipDeviceReset();
        return -1;
    }

    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error after synchronization: %s\n", hipGetErrorString(err));
        hipDeviceReset();
        return -1;
    }

    // Copy timing data back to host
    hipMemcpy(h_timings, d_timings, totalThreads * sizeof(ThreadTiming), hipMemcpyDeviceToHost);

    // Write timing data to CSV
    write_timing_to_csv("thread_timing.csv", h_timings, totalThreads);

    // Check if address was found
    hipMemcpyFromSymbol(&h_address_found, HIP_SYMBOL(d_address_found), sizeof(bool));
    if (h_address_found) {
        hipMemcpyFromSymbol(h_address_value, HIP_SYMBOL(d_address_value), P_CHAIN_ADDRESS_LENGTH + 1);
        printf("\nAddress found: %s\n", h_address_value);
        hipMemcpyFromSymbol(h_passphrase_value, HIP_SYMBOL(d_passphrase_value), MAX_PASSPHRASE_LENGTH);
        printf("Passphrase: %s\n", h_passphrase_value);

        // Save results to file
        std::ofstream result_file("result.txt");
        if (result_file.is_open()) {
            result_file << "Address: " << h_address_value << std::endl;
            result_file << "Passphrase: " << h_passphrase_value << std::endl;
            result_file.close();
            std::cout << "Results saved to result.txt" << std::endl;
        } else {
            std::cerr << "Unable to open result.txt for writing" << std::endl;
        }
    } else {
        printf("\nAddress not found\n");
    }

    // Clean up
    hipFree(d_expected_value);
    hipFree(d_mnemonic);
    hipDeviceReset();

    delete[] h_timings;
    hipFree(d_timings);

    return 0;
}