#include "hip/hip_runtime.h"
#include <fstream>
#include <iomanip>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"
#include "pbkdf2.h"

#define TEST_BIGNUM_WORDS 4

__device__ void print_as_hex_char_tmp(unsigned char *data, int len) {
    for (int i = 0; i < len; i++) {
        printf("%02x", data[i]);
    }
    printf("\n");
}

// BIP32 ++
typedef struct {
    unsigned char master_private_key[32];
    unsigned char chain_code[32];
} BIP32Info;

__device__ void my_cuda_memcpy_unsigned_char_b(uint8_t *dst, const uint8_t *src, unsigned int n) {
    for (unsigned int i = 0; i < n; ++i) {
        dst[i] = src[i];
    }
}

__device__ BIP32Info bip32_from_seed_kernel(const uint8_t *seed, uint32_t seed_len) {
    printf("++ bip32_from_seed_kernel ++\n");
    printf(">> seed: ");
    print_as_hex(seed, seed_len);
    printf(">> seed_len: %d\n", seed_len);

    BIP32Info info;
	// Initialize HMAC_SHA512_CTX
    HMAC_SHA512_CTX hmac;
    
    // Compute HMAC-SHA512 with "Bitcoin seed" as the key
    hmac_sha512_init(&hmac, (const uint8_t *)"Bitcoin seed", 12);
    hmac_sha512_update(&hmac, seed, seed_len);

    // Print hmac
    // printf("# hmac: ");
    
    unsigned char hash[64];
    // clear hash
    for (int i = 0; i < 64; ++i) {
        hash[i] = 0;
    }
    hmac_sha512_final(&hmac, hash);

    // Print hash
    printf("# hash: ");
    print_as_hex(hash, 64);
    
    // Copy the first 32 bytes to master_private_key and the next 32 bytes to chain_code
    //my_cuda_memcpy_unsigned_char(info->master_private_key, hash, 32);
    //my_cuda_memcpy_unsigned_char(info->chain_code, hash + 32, 32);
	my_cuda_memcpy_unsigned_char_b(info.master_private_key, hash, 32);
	my_cuda_memcpy_unsigned_char_b(info.chain_code, hash + 32, 32);

    printf("-- bip32_from_seed_kernel --\n");
	return info;
}
// BIP32 --

// Public key derivation ++
// __device__ void derive_public_key(BIGNUM* private_key, BIGNUM* publicKey) {
// 	// point_mul(private_key, publicKey);
//     printf("++ derive_public_key ++\n");
//     bn_print(">> private: ", private_key);
//     bn_print(">> public: ", publicKey);
//     printf("-- derive_public_key --\n");
// }
// Public key derivation --

// Child key derivation ++
__device__ void my_cuda_memcpy_uint32_t(uint32_t *dst, const uint32_t *src, unsigned int n) {
    for (unsigned int i = 0; i < n / sizeof(uint32_t); ++i) {
        uint32_t val = src[i];
        dst[i] = __byte_perm(val, 0, 0x0123);
    }
}

__device__ void my_cuda_memcpy_uint32_t_to_unsigned_char(unsigned char *dst, const uint32_t *src, unsigned int n) {
    for (unsigned int i = 0; i < n / sizeof(uint32_t); ++i) {
        uint32_t val = src[i];
        dst[4 * i] = (val) & 0xFF;
        dst[4 * i + 1] = (val >> 8) & 0xFF;
        dst[4 * i + 2] = (val >> 16) & 0xFF;
        dst[4 * i + 3] = (val >> 24) & 0xFF;
    }
}

__device__ BIP32Info GetChildKeyDerivation(uint8_t* key, uint8_t* chainCode, uint32_t index) {
	printf("++ GetChildKeyDerivation ++\n");
    printf(">> key: ");
    print_as_hex(key, 32);
    printf(">> chainCode: ");
    print_as_hex(chainCode, 32);
    printf(">> index: %u\n", index);
    printf("\n* step 0 index: %u\n", index);
    BIP32Info info;

    // Compute HMAC-SHA512
    HMAC_SHA512_CTX hmac;
    uint8_t buffer[100];
    uint8_t hash[64];
    unsigned int len = 64;

    // Fill buffer according to index
    if (index == 0) {
        // TODO: Generate the public key from the parent private key and store it in buffer
		// printf("!!! Public key generation not implemented yet !!!\n");
		printf("    * INDEX is 0\n");
		// size_t publicKeyLen = 0;
		// unsigned char *publicKeyBytes = GetPublicKey(key, 32, &publicKeyLen);
		// print_as_hex_char(publicKeyBytes, publicKeyLen);
		// memcpy(buffer, publicKeyBytes, 33);  // Copies the entire 33-byte compressed public key including the first byte
		// buffer_len += 33;
        
        BIGNUM newKey;
        init_zero(&newKey);
        for (int i = 0; i < 4; ++i) {
            newKey.d[3 - i] = ((BN_ULONG)key[8*i] << 56) | 
                              ((BN_ULONG)key[8*i + 1] << 48) | 
                              ((BN_ULONG)key[8*i + 2] << 40) | 
                              ((BN_ULONG)key[8*i + 3] << 32) |
                              ((BN_ULONG)key[8*i + 4] << 24) | 
                              ((BN_ULONG)key[8*i + 5] << 16) | 
                              ((BN_ULONG)key[8*i + 6] << 8) | 
                              ((BN_ULONG)key[8*i + 7]);
        }
        printf("      * Cuda newKey:");
        bn_print("", &newKey);
        
        // Initialize constants //TODO: Move it outside of each THREAD. Call once before instead and then sync
        init_zero(&CURVE_A);
        
        // For secp256k1, CURVE_B should be initialized to 7 rather than 0
        init_zero(&CURVE_B);
        CURVE_B.d[0] = 0x7;

        BN_ULONG CURVE_GX_values[MAX_BIGNUM_SIZE] = {
            0x79BE667EF9DCBBAC,
            0x55A06295CE870B07,
            0x029BFCDB2DCE28D9,
            0x59F2815B16F81798
            };
        for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
                CURVE_GX_d[j] = CURVE_GX_values[j];
            }

        // Generator y coordinate
        // BIGNUM CURVE_GY;
        BN_ULONG CURVE_GY_values[MAX_BIGNUM_SIZE] = {
            0x483ADA7726A3C465,
            0x5DA4FBFC0E1108A8,
            0xFD17B448A6855419,
            0x9C47D08FFB10D4B8
            };
        for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
                CURVE_GY_d[j] = CURVE_GY_values[j];
            }

        // Initialize generator
        EC_POINT G;
        init_zero(&G.x);
        init_zero(&G.y);
        for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
                G.x.d[j] = CURVE_GX_values[j];
                G.y.d[j] = CURVE_GY_values[j];
            }
        // reverse
        reverse_order(&G.x, TEST_BIGNUM_WORDS);
        reverse_order(&G.y, TEST_BIGNUM_WORDS);
        // find top
        G.x.top = find_top(&G.x);
        G.y.top = find_top(&G.y);

        init_zero(&CURVE_P);
        // Init curve prime
        // fffffffffffffffffffffffffffffffffffffffffffffffffffffffefffffc2f
        BN_ULONG CURVE_P_values[MAX_BIGNUM_SIZE] = {
            0xFFFFFFFFFFFFFFFF,
            0xFFFFFFFFFFFFFFFF,
            0xFFFFFFFFFFFFFFFF,
            0xFFFFFFFEFFFFFC2F,
            0,0,0,0        
            };
        for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
                CURVE_P.d[j] = CURVE_P_values[j];
            }
        // reverse
        reverse_order(&CURVE_P, TEST_BIGNUM_WORDS);
        // find top
        CURVE_P.top = find_top(&CURVE_P);
        // TODO: Check do we need to define curves, G and do reversing
        EC_POINT publicKey = ec_point_scalar_mul(&G, &newKey, &CURVE_P, &CURVE_A);
        // print &publicKey.x
        printf("      * Cuda publicKey.x: ");
        bn_print("", &publicKey.x);
        // print &publicKey.y
        printf("      * Cuda publicKey.y: ");
        bn_print("", &publicKey.y);

        return info; // TODO: Get 03 concatenated to publicKey.x as buffer

    } else {
        buffer[0] = 0;
        my_cuda_memcpy_unsigned_char(buffer + 1, key, 32);
    }

    // Append index in big-endian format to buffer
    buffer[33] = (index >> 24) & 0xFF;
    buffer[34] = (index >> 16) & 0xFF;
    buffer[35] = (index >> 8) & 0xFF;
    buffer[36] = index & 0xFF;

	hmac_sha512_init(&hmac, chainCode, 32);
    hmac_sha512_update(&hmac, buffer, 37);  // Assuming buffer_len = 37 // TODO: Check would it be defined in "int len"? 64
    hmac_sha512_final(&hmac, hash);

	// Print the pre-HMAC values
    printf("      * Cuda Pre-HMAC variable key:");
    for (int i = 0; i < 32; i++) {
        printf("%02x", key[i]);
    }
    printf("\n");

    printf("      * Cuda Pre-HMAC Buffer:");
    for (int i = 0; i < 37; i++) { // Assuming the buffer length up to the index is 37
        printf("%02x", buffer[i]);
    }
    printf("\n");

    printf("      * Cuda Pre-HMAC Key:");
    for (int i = 0; i < 32; i++) {
        printf("%02x", chainCode[i]);
    }
    printf("\n");   

	uint32_t il[8], ir[8];
	
	// Populate il and ir from hash
	my_cuda_memcpy_uint32_t(il, (uint32_t*)hash, 8 * sizeof(uint32_t)); // Using uint32_t version for il
	my_cuda_memcpy_uint32_t(ir, (uint32_t*)(hash + 32), 8 * sizeof(uint32_t)); // Using uint32_t version for ir

    // Print the hash from 32 to 64
    printf("      * Cuda hash from 32 to 64:");
    for (int i = 32; i < 64; i++) {
        printf("%02x", hash[i]);
    }
    printf("\n");

    // Copy the hash (from 32 to 64) to chain_code
    my_cuda_memcpy_unsigned_char(info.chain_code, hash + 32, 32);

    // ***

    // return info;

	// After HMAC-SHA512
	printf("      * Cuda Post-HMAC hash:");
	for (int i = 0; i < 64; i++) {
		printf("%02x", hash[i]);
	}
	printf("\n");

	printf("      * Cuda il as uint32_t: ");
	for (int i = 0; i < 8; ++i) {
		printf("%08x", il[i]);
	}
	printf("\n");

	printf("      * Cuda ir as uint32_t: ");
	for (int i = 0; i < 8; ++i) {
		printf("%08x", ir[i]);
	}
	printf("\n");

    printf("      * Cuda ir as uint64_t: ");
    uint64_t ir_64[4];
    for (int i = 0; i < 8; ++i) {
        ir_64[i] = ((uint64_t)ir[2*i] << 32) | (uint64_t)ir[2*i + 1];
    }
    for (int i = 0; i < 4; ++i) {
        printf("%016lx", ir_64[i]);
    }
    printf("\n");
    

	// Print individual bytes of ir before copying
	// printf("      * Individual bytes of Cuda ir before copying: ");
	// uint8_t *ir_bytes = (uint8_t *) ir;
	// for (int i = 0; i < 32; ++i) {
	// 	printf("%02x", ir_bytes[i]);
	// }
	// printf("\n");

	// Perform the copy
	// my_cuda_memcpy_uint32_t_to_unsigned_char(info.chain_code, ir, 32);
    // Copy ir_64 to chain_code
    for (int i = 0; i < 4; i++) {
        info.chain_code[8*i] = (ir_64[i] >> 56) & 0xFF;
        info.chain_code[8*i + 1] = (ir_64[i] >> 48) & 0xFF;
        info.chain_code[8*i + 2] = (ir_64[i] >> 40) & 0xFF;
        info.chain_code[8*i + 3] = (ir_64[i] >> 32) & 0xFF;
        info.chain_code[8*i + 4] = (ir_64[i] >> 24) & 0xFF;
        info.chain_code[8*i + 5] = (ir_64[i] >> 16) & 0xFF;
        info.chain_code[8*i + 6] = (ir_64[i] >> 8) & 0xFF;
        info.chain_code[8*i + 7] = ir_64[i] & 0xFF;
    }

	// Print individual bytes of chain_code after copying
	printf("      * Individual bytes of Cuda chain_code after copying: ");
	for (int i = 0; i < 32; ++i) {
		printf("%02x", info.chain_code[i]);
	}
	printf("\n");

	// After populating il and ir
	printf("    * il: ");
	for (int i = 0; i < 8; i++) {
		printf("%08x", il[i]);
	}
	printf("\n");
	printf("    * ir: ");
	for (int i = 0; i < 8; i++) {
		printf("%08x", ir[i]);
	}
	printf("\n");

    // ir is uint32_t[8]
    // info.chain_code is unsigned char[32]
    
	
	// Addition
	BIGNUM a;
	BIGNUM b;
	BIGNUM curveOrder;
	BIGNUM newKey;
	BIGNUM publicKey;

    init_zero(&a);
    init_zero(&b);
    init_zero(&curveOrder);
    init_zero(&newKey);
    init_zero(&publicKey);

	BN_ULONG a_d[8];
  	BN_ULONG b_d[8];
	BN_ULONG newKey_d[8];
  	// BN_ULONG curveOrder_d[16];
	BN_ULONG publicKey_d[8];
	// uint32_t curveOrder[8] = {0xffffffff, 0xffffffff, 0xffffffff, 0xfffffffe, 0xbaaedce6, 0xaf48a03b, 0xbfd25e8c, 0xd0364141};
	// Initialize curveOrder_d for secp256k1
	// FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141
    // 
    curveOrder.d[0] = 0xBFD25E8CD0364141;
    curveOrder.d[1] = 0xBAAEDCE6AF48A03B;
    curveOrder.d[2] = 0xFFFFFFFFFFFFFFFE;
    curveOrder.d[3] = 0xFFFFFFFFFFFFFFFF;
    curveOrder.neg = 0;
    curveOrder.top = 4;
    
    // hash: uint8_t[64]
    // il: uint32_t il[8]
    // a.d: is BN_ULONG
    // Initialize a from il
    for (int i = 0; i < 4; ++i) {
        a.d[3 - i] = ((BN_ULONG)il[2*i] << 32) | (BN_ULONG)il[2*i + 1];
    }
    a.neg = 0;
    a.top = 4;  // We're using 4 64-bit words
    bn_print("A: ", &a);

	// key: uint8_t*
    // b.d: BN_ULONG
    // Initialize b from key
	for (int i = 0; i < 4; ++i) {
        b.d[3 - i] = ((BN_ULONG)key[8*i] << 56) | 
                     ((BN_ULONG)key[8*i + 1] << 48) | 
                     ((BN_ULONG)key[8*i + 2] << 40) | 
                     ((BN_ULONG)key[8*i + 3] << 32) |
                     ((BN_ULONG)key[8*i + 4] << 24) | 
                     ((BN_ULONG)key[8*i + 5] << 16) | 
                     ((BN_ULONG)key[8*i + 6] << 8) | 
                     ((BN_ULONG)key[8*i + 7]);
    }
    b.neg = 0;
    b.top = 4;  // We're using 4 64-bit words
    bn_print("B: ", &b);

    // return info;

	// Initialize newKey_d
	for (int i = 0; i < 8; i++) newKey_d[i] = 0;
	// newKey.d = newKey_d;
    for (int j = 0; j < 8; ++j) {
        newKey.d[j] = newKey_d[j]; // TODO: Check do we need to reverse the order
    }
	newKey.neg = 0;
	// newKey.top = 8;
    newKey.top = find_top(&newKey);
    bn_print("Debug Cuda newKey (Before add): ", &newKey);
	
    bn_add(&newKey, &a, &b);

    // Print A + B
    bn_print("Debug Cuda newKey (After add): ", &newKey);

    // Print curve order
    bn_print("Debug Cuda curveOrder: ", &curveOrder);

    printf("Calling bn_mod\n");
    bn_mod(&newKey, &newKey, &curveOrder);

    // printf("After bn_mod\n");
    bn_print("Debug Cuda newKey (After mod): ", &newKey);

    // Copy newKey to info.master_private_key
    // for (int i = 0; i < 8; i++) {
    //     info.master_private_key[i] = newKey.d[i];
    // }
    // Copy newKey to info.master_private_key
    for (int i = 0; i < 4; i++) {
        info.master_private_key[8*i] = (newKey.d[3 - i] >> 56) & 0xFF;
        info.master_private_key[8*i + 1] = (newKey.d[3 - i] >> 48) & 0xFF;
        info.master_private_key[8*i + 2] = (newKey.d[3 - i] >> 40) & 0xFF;
        info.master_private_key[8*i + 3] = (newKey.d[3 - i] >> 32) & 0xFF;
        info.master_private_key[8*i + 4] = (newKey.d[3 - i] >> 24) & 0xFF;
        info.master_private_key[8*i + 5] = (newKey.d[3 - i] >> 16) & 0xFF;
        info.master_private_key[8*i + 6] = (newKey.d[3 - i] >> 8) & 0xFF;
        info.master_private_key[8*i + 7] = newKey.d[3 - i] & 0xFF;
    }
	
    printf("\n");
    return info;

	// bn_print("  * private: ", &newKey);
	// printf("\n");

	// // uint8_t newKeyBytes[32] = {0};  // Initialize to zero
	// printf("\n");
	// printf("  * public: ");	
	// size_t publicKeyLen = 0;
	// // Initialize public key
	// // BIGNUM publicKey;
	// for (int i = 0; i < 8; i++) publicKey_d[i] = 0;
	// // publicKey.d = publicKey_d;
    // for (int j = 0; j < 8; ++j) {
    //     publicKey.d[j] = publicKey_d[j]; // TODO: Check do we need to reverse the order
    // }
	// publicKey.neg = 0;
	// publicKey.top = 0;

	// // getPublicKey(&newKey, &publicKey, &publicKeyLen);
	// // Derive public key
    // derive_public_key(&newKey, &publicKey);

	// // Print the public key
	// for (int i = 0; i < 8; i++) {
	// 	printf("%02x", publicKey.d[i]);
	// }
	// printf("\n");

    // return info;
}
// Child key derivation --

// __device__ void reverse_order(BIGNUM *test_values_a) {
//     for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
//         BN_ULONG temp_a = test_values_a->d[j];
//         test_values_a->d[j] = test_values_a->d[TEST_BIGNUM_WORDS - 1 - j];
//         test_values_a->d[TEST_BIGNUM_WORDS - 1 - j] = temp_a;
//     }
// }

__global__ void search_kernel() {
    printf("++ search_kernel ++\n");

    // Convert the mnemonic and passphrase to byte arrays
    uint8_t *m_mnemonic = (unsigned char *)"sell stereo useless course suffer tribe jazz monster fresh excess wire again father film sudden pelican always room attack rubber pelican trash alone cancel";
    // print as hex
    print_as_hex(m_mnemonic, 156);

    uint8_t *salt = (unsigned char *)"mnemonicTESTPHRASE";
    unsigned char bip39seed[64];  // This will hold the generated seed
    // Initialize bip39seed to zeros
    for (int i = 0; i < 64; ++i) {
        bip39seed[i] = 0;
    }

    // Call pbkdf2_hmac to perform the bip39seed key derivation
    compute_pbkdf2(
        (uint8_t *) m_mnemonic, 
        my_strlen((const char*) m_mnemonic), 
        (uint8_t *) salt, 
        my_strlen((const char*) salt),
	    2048, 
        64,
        bip39seed
        );
    printf("bip39seed: ");
    print_as_hex(bip39seed, 64);

    // Bip32FromSeed
    BIP32Info master_key = bip32_from_seed_kernel(bip39seed, 64);
    printf("\nMaster Chain Code: ");
    print_as_hex_char_tmp(master_key.chain_code, 32);
    printf("\nMaster Private Key: ");
    print_as_hex_char_tmp(master_key.master_private_key, 32);
    
    // Child key derivation
	uint32_t index44 = 0x8000002C;
	uint32_t index9000 = 0x80002328;
	uint32_t index0Hardened = 0x80000000;
	uint32_t index0 = 0x00000000;
    // TODO: remove _index from child_key variable. Write to the same variable instead.
	BIP32Info child_key = GetChildKeyDerivation(master_key.master_private_key, master_key.chain_code, index44);
	printf("[0] Child Chain Code: ");
	print_as_hex_char_tmp(child_key.chain_code, 32);
	printf("[0] Child Private Key: ");
	print_as_hex_char_tmp(child_key.master_private_key, 32);
    
    child_key = GetChildKeyDerivation(child_key.master_private_key, child_key.chain_code, index9000);
    printf("[1] Child Chain Code: ");
    print_as_hex_char_tmp(child_key.chain_code, 32);
    printf("[1] Child Private Key: ");
    print_as_hex_char_tmp(child_key.master_private_key, 32);

    child_key = GetChildKeyDerivation(child_key.master_private_key, child_key.chain_code, index0Hardened);
    printf("[2] Child Chain Code: ");
    print_as_hex_char_tmp(child_key.chain_code, 32);
    printf("[2] Child Private Key: ");
    print_as_hex_char_tmp(child_key.master_private_key, 32);

    child_key = GetChildKeyDerivation(child_key.master_private_key, child_key.chain_code, index0);
    printf("[3] Child Chain Code: ");
    print_as_hex_char_tmp(child_key.chain_code, 32);
    printf("[3] Child Private Key: ");
    print_as_hex_char_tmp(child_key.master_private_key, 32);

    printf("\n-- search_kernel --\n");    
}

int main() {
    
    const int THREADS_PER_BLOCK = 1;
    // const int THREADS_PER_BLOCK = 256; // A good balance between occupancy and flexibility
    
    const int NUM_BLOCKS = 1;
    // const int NUM_BLOCKS = 128; // One block per SM OK

    // Launch kernel
    search_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>();

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}