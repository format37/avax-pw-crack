// ++ Bip39SeedGenerator ++

#include <hip/hip_runtime.h>
#include <cstdint>

// The rotate operation for 64bits
#define ROR64(x,n) ((x >> n) | (x << (64 - n)))
#define CH(x,y,z)  (z ^ (x & (y ^ z)))
#define MAJ(x,y,z) ((x & y) | (z & (x | y)))
#define S0_64(x)   (ROR64((x), 28) ^ ROR64((x),  34) ^ ROR64((x), 39)) 
#define S1_64(x)   (ROR64((x), 14) ^ ROR64((x),  18) ^ ROR64((x), 41)) 
#define R0_64(x)   (ROR64((x), 1)  ^ ROR64((x),  8)  ^ ((x) >> 7)) 
#define R1_64(x)   (ROR64((x), 19) ^ ROR64((x), 61) ^ ((x) >> 6))

#ifndef PBKDF2_SHA512_INCLUDE
#define PBKDF2_SHA512_INCLUDE

#define SHA512_BLOCKLEN  128ul
#define SHA512_DIGESTLEN 64ul
#define SHA512_DIGESTINT 8ul

#ifndef PBKDF2_SHA512_STATIC
#define PBKDF2_SHA512_DEF extern
#else
#define PBKDF2_SHA512_DEF static
#endif
#include <string.h>
// -- Bip39SeedGenerator --

// ++ Child key derivation ++
#include <stdio.h>
#include <stdlib.h>
#define PBKDF2_SHA512_STATIC
#define PBKDF2_SHA512_IMPLEMENTATION
#include <openssl/evp.h>
#include <openssl/crypto.h>
#include <openssl/hmac.h>
#include <openssl/bn.h>
#include <openssl/ec.h>
#include <openssl/obj_mac.h>
//#include <openssl/sha.h>
#include <openssl/ripemd.h>
//#include <openssl/sha.h>
#define MY_SHA256_DIGEST_LENGTH 32
#define CHARSET "qpzry9x8gf2tvdw0s3jn54khce6mua7l"
#define CHECKSUM_LENGTH 6
// -- Child key derivation --



// ++ Bip39SeedGenerator ++
typedef struct sha512_ctx_t
{
    uint64_t len;  // Make sure this is uint64_t
    uint64_t h[SHA512_DIGESTINT];
    uint8_t buf[SHA512_BLOCKLEN];
} SHA512_CTX;

__device__ void sha512_init(SHA512_CTX *ctx);
__device__ void sha512_update(SHA512_CTX *ctx, const uint8_t *m, uint32_t mlen);
__device__ void sha512_final(SHA512_CTX *ctx, uint8_t *md);

typedef struct hmac_sha512_ctx_t
{
	uint8_t buf[SHA512_BLOCKLEN]; // key block buffer, not needed after init
	uint64_t h_inner[SHA512_DIGESTINT];
	uint64_t h_outer[SHA512_DIGESTINT];
	SHA512_CTX sha;
} HMAC_SHA512_CTX;

PBKDF2_SHA512_DEF __device__ void hmac_sha512_init(HMAC_SHA512_CTX *hmac, const uint8_t *key, uint32_t keylen);
PBKDF2_SHA512_DEF __device__ void hmac_sha512_update(HMAC_SHA512_CTX *hmac, const uint8_t *m, uint32_t mlen);
// resets state to hmac_sha512_init
PBKDF2_SHA512_DEF __device__ void hmac_sha512_final(HMAC_SHA512_CTX *hmac, uint8_t *md);

PBKDF2_SHA512_DEF __device__ void pbkdf2_sha512(HMAC_SHA512_CTX *ctx,
    const uint8_t *key, uint32_t keylen, const uint8_t *salt, uint32_t saltlen, uint32_t rounds,
    uint8_t *dk, uint32_t dklen);

#endif // PBKDF2_SHA512_INCLUDE

//------------------------------------------------------------------------------

#ifdef PBKDF2_SHA512_IMPLEMENTATION

#include <string.h>

#define ROR(n,k) ror(n,k)

#define CH(x,y,z)  (z ^ (x & (y ^ z)))
#define MAJ(x,y,z) ((x & y) | (z & (x | y)))
#define S0(x)      (ROR(x, 2) ^ ROR(x,13) ^ ROR(x,22))
#define S1(x)      (ROR(x, 6) ^ ROR(x,11) ^ ROR(x,25))
#define R0(x)      (ROR(x, 7) ^ ROR(x,18) ^ (x>>3))
#define R1(x)      (ROR(x,17) ^ ROR(x,19) ^ (x>>10))

#endif

#define INNER_PAD '\x36'
#define OUTER_PAD '\x5c'

__device__ static const uint64_t K[80] = {
    UINT64_C(0x428a2f98d728ae22), UINT64_C(0x7137449123ef65cd),
    UINT64_C(0xb5c0fbcfec4d3b2f), UINT64_C(0xe9b5dba58189dbbc),
    UINT64_C(0x3956c25bf348b538), UINT64_C(0x59f111f1b605d019),
    UINT64_C(0x923f82a4af194f9b), UINT64_C(0xab1c5ed5da6d8118),
    UINT64_C(0xd807aa98a3030242), UINT64_C(0x12835b0145706fbe),
    UINT64_C(0x243185be4ee4b28c), UINT64_C(0x550c7dc3d5ffb4e2),
    UINT64_C(0x72be5d74f27b896f), UINT64_C(0x80deb1fe3b1696b1),
    UINT64_C(0x9bdc06a725c71235), UINT64_C(0xc19bf174cf692694),
    UINT64_C(0xe49b69c19ef14ad2), UINT64_C(0xefbe4786384f25e3),
    UINT64_C(0x0fc19dc68b8cd5b5), UINT64_C(0x240ca1cc77ac9c65),
    UINT64_C(0x2de92c6f592b0275), UINT64_C(0x4a7484aa6ea6e483),
    UINT64_C(0x5cb0a9dcbd41fbd4), UINT64_C(0x76f988da831153b5),
    UINT64_C(0x983e5152ee66dfab), UINT64_C(0xa831c66d2db43210),
    UINT64_C(0xb00327c898fb213f), UINT64_C(0xbf597fc7beef0ee4),
    UINT64_C(0xc6e00bf33da88fc2), UINT64_C(0xd5a79147930aa725),
    UINT64_C(0x06ca6351e003826f), UINT64_C(0x142929670a0e6e70),
    UINT64_C(0x27b70a8546d22ffc), UINT64_C(0x2e1b21385c26c926),
    UINT64_C(0x4d2c6dfc5ac42aed), UINT64_C(0x53380d139d95b3df),
    UINT64_C(0x650a73548baf63de), UINT64_C(0x766a0abb3c77b2a8),
    UINT64_C(0x81c2c92e47edaee6), UINT64_C(0x92722c851482353b),
    UINT64_C(0xa2bfe8a14cf10364), UINT64_C(0xa81a664bbc423001),
    UINT64_C(0xc24b8b70d0f89791), UINT64_C(0xc76c51a30654be30),
    UINT64_C(0xd192e819d6ef5218), UINT64_C(0xd69906245565a910),
    UINT64_C(0xf40e35855771202a), UINT64_C(0x106aa07032bbd1b8),
    UINT64_C(0x19a4c116b8d2d0c8), UINT64_C(0x1e376c085141ab53),
    UINT64_C(0x2748774cdf8eeb99), UINT64_C(0x34b0bcb5e19b48a8),
    UINT64_C(0x391c0cb3c5c95a63), UINT64_C(0x4ed8aa4ae3418acb),
    UINT64_C(0x5b9cca4f7763e373), UINT64_C(0x682e6ff3d6b2b8a3),
    UINT64_C(0x748f82ee5defb2fc), UINT64_C(0x78a5636f43172f60),
    UINT64_C(0x84c87814a1f0ab72), UINT64_C(0x8cc702081a6439ec),
    UINT64_C(0x90befffa23631e28), UINT64_C(0xa4506cebde82bde9),
    UINT64_C(0xbef9a3f7b2c67915), UINT64_C(0xc67178f2e372532b),
    UINT64_C(0xca273eceea26619c), UINT64_C(0xd186b8c721c0c207),
    UINT64_C(0xeada7dd6cde0eb1e), UINT64_C(0xf57d4f7fee6ed178),
    UINT64_C(0x06f067aa72176fba), UINT64_C(0x0a637dc5a2c898a6),
    UINT64_C(0x113f9804bef90dae), UINT64_C(0x1b710b35131c471b),
    UINT64_C(0x28db77f523047d84), UINT64_C(0x32caab7b40c72493),
    UINT64_C(0x3c9ebe0a15c9bebc), UINT64_C(0x431d67c49c100d4c),
    UINT64_C(0x4cc5d4becb3e42b6), UINT64_C(0x597f299cfc657e2a),
    UINT64_C(0x5fcb6fab3ad6faec), UINT64_C(0x6c44198c4a475817),
};

__device__ void my_cuda_memcpy_uint64(uint64_t *dst, const uint64_t *src, unsigned int n) {
    for (unsigned int i = 0; i < n / sizeof(uint64_t); ++i) {  // assuming n is in bytes
        dst[i] = src[i];
    }
}

__device__ void my_cuda_memcpy_unsigned_char(uint8_t *dst, const uint8_t *src, unsigned int n) {
    for (unsigned int i = 0; i < n; ++i) {
        dst[i] = src[i];
    }
}

__device__ size_t my_strlen(const char *str) {
    size_t len = 0;
    while (*str != '\0') {
        ++len;
        ++str;
    }
    return len;
}

__device__ void print_as_hex(const uint8_t *s,  const uint32_t slen)
{
	for (uint32_t i = 0; i < slen; i++)
	{
		printf("%02X%s", s[ i ], (i % 4 == 3) && (i != slen - 1) ? "-" : "");
	}
	printf("\n");
}

__device__ void sha512_init(SHA512_CTX *s)
{
	s->len = 0;
	s->h[0] = 0x6a09e667f3bcc908ULL;
	s->h[1] = 0xbb67ae8584caa73bULL;
	s->h[2] = 0x3c6ef372fe94f82bULL;
	s->h[3] = 0xa54ff53a5f1d36f1ULL;
	s->h[4] = 0x510e527fade682d1ULL;
	s->h[5] = 0x9b05688c2b3e6c1fULL;
	s->h[6] = 0x1f83d9abfb41bd6bULL;
	s->h[7] = 0x5be0cd19137e2179ULL;
}

__device__ static void sha512_transform(SHA512_CTX *s, const uint8_t *buf)
{
    uint64_t t1, t2, a, b, c, d, e, f, g, h, m[80];
    uint32_t i, j;

    for (i = 0, j = 0; i < 16; i++, j += 8)
    {
        m[i] = ((uint64_t)buf[j] << 56) | ((uint64_t)buf[j + 1] << 48) |
               ((uint64_t)buf[j + 2] << 40) | ((uint64_t)buf[j + 3] << 32) |
               ((uint64_t)buf[j + 4] << 24) | ((uint64_t)buf[j + 5] << 16) |
               ((uint64_t)buf[j + 6] << 8) | ((uint64_t)buf[j + 7]);
    }
    for (; i < 80; i++)
    {
        m[i] = R1_64(m[i - 2]) + m[i - 7] + R0_64(m[i - 15]) + m[i - 16];
    }

    a = s->h[0];
    b = s->h[1];
    c = s->h[2];
    d = s->h[3];
    e = s->h[4];
    f = s->h[5];
    g = s->h[6];
    h = s->h[7];

    for (i = 0; i < 80; i++) // Increase loop limit to 80
    {
        t1 = h + S1_64(e) + CH(e, f, g) + K[i] + m[i];
        t2 = S0_64(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
    }

	s->h[0] += a;
	s->h[1] += b;
	s->h[2] += c;
	s->h[3] += d;
	s->h[4] += e;
	s->h[5] += f;
	s->h[6] += g;
	s->h[7] += h;
}

__device__ void sha512_update(SHA512_CTX *s, const uint8_t *m, uint32_t len)
{
	const uint8_t *p = m;
	uint32_t r = s->len % SHA512_BLOCKLEN;
	
	s->len += len;
	if (r)
	{
		if (len + r < SHA512_BLOCKLEN)
		{
            my_cuda_memcpy_unsigned_char(s->buf + r, p, len);
			return;
		}
        my_cuda_memcpy_unsigned_char(s->buf + r, p, SHA512_BLOCKLEN - r);
		len -= SHA512_BLOCKLEN - r;
		p += SHA512_BLOCKLEN - r;
		sha512_transform(s, s->buf);
	}
	for (; len >= SHA512_BLOCKLEN; len -= SHA512_BLOCKLEN, p += SHA512_BLOCKLEN)
	{
		sha512_transform(s, p);
	}
    my_cuda_memcpy_unsigned_char(s->buf, p, len);
}

__device__ void sha512_final(SHA512_CTX *s, uint8_t *md)
{
	uint32_t r = s->len % SHA512_BLOCKLEN;
	uint64_t totalBits = s->len * 8;  // Total bits
	uint64_t len_lower = totalBits & 0xFFFFFFFFFFFFFFFFULL;  // Lower 64 bits
    uint64_t len_upper = 0;  // Upper 64 bits are zero for 64-bit totalBits

	
    // Pad message
    s->buf[r++] = 0x80;
    while (r < 112)  // Padding until the total length is 112
    {
        s->buf[r++] = 0x00;
    }

    // Write 128 bit processed length in big-endian
    for (int i = 0; i < 8; ++i)
    {
		s->buf[r++] = (len_upper >> (8 * (7 - i))) & 0xFF;
	}

	for (int i = 0; i < 8; ++i)
    {
		s->buf[r++] = (len_lower >> (8 * (7 - i))) & 0xFF;
	}
	sha512_transform(s, s->buf);
	
	for (uint32_t i = 0; i < SHA512_DIGESTINT; i++)
	{
		md[8 * i    ] = s->h[i] >> 56;
		md[8 * i + 1] = s->h[i] >> 48;
		md[8 * i + 2] = s->h[i] >> 40;
		md[8 * i + 3] = s->h[i] >> 32;
		md[8 * i + 4] = s->h[i] >> 24;
		md[8 * i + 5] = s->h[i] >> 16;
		md[8 * i + 6] = s->h[i] >> 8;
		md[8 * i + 7] = s->h[i];
	}
	sha512_init(s);
}

PBKDF2_SHA512_DEF __device__ void hmac_sha512_init(HMAC_SHA512_CTX *hmac, const uint8_t *key, uint32_t keylen)
{
	SHA512_CTX *sha = &hmac->sha;
	
	if (keylen <= SHA512_BLOCKLEN)
	{
        my_cuda_memcpy_unsigned_char(hmac->buf, key, keylen);
		memset(hmac->buf + keylen, '\0', SHA512_BLOCKLEN - keylen);
	}
	else
	{
		sha512_init(sha);
		sha512_update(sha, key, keylen);
		sha512_final(sha, hmac->buf);
		memset(hmac->buf + SHA512_DIGESTLEN, '\0', SHA512_BLOCKLEN - SHA512_DIGESTLEN);
	}
	
	uint32_t i;
	for (i = 0; i < SHA512_BLOCKLEN; i++)
	{
		hmac->buf[ i ] = hmac->buf[ i ] ^ OUTER_PAD;
	}
	sha512_init(sha);
	sha512_update(sha, hmac->buf, SHA512_BLOCKLEN);
	// copy outer state
    my_cuda_memcpy_uint64(hmac->h_outer, sha->h, SHA512_DIGESTLEN);	
	for (i = 0; i < SHA512_BLOCKLEN; i++)
	{
		hmac->buf[ i ] = (hmac->buf[ i ] ^ OUTER_PAD) ^ INNER_PAD;
	}
	
	sha512_init(sha);
	sha512_update(sha, hmac->buf, SHA512_BLOCKLEN);
	// copy inner state
    my_cuda_memcpy_uint64(hmac->h_inner, sha->h, SHA512_DIGESTLEN);
}

PBKDF2_SHA512_DEF __device__ void hmac_sha512_update(HMAC_SHA512_CTX *hmac, const uint8_t *m, uint32_t mlen)
{
	sha512_update(&hmac->sha, m, mlen);
}

PBKDF2_SHA512_DEF __device__ void hmac_sha512_final(HMAC_SHA512_CTX *hmac, uint8_t *md)
{
	SHA512_CTX *sha = &hmac->sha;
	sha512_final(sha, md);
	
	// reset sha to outer state
    my_cuda_memcpy_uint64(sha->h, hmac->h_outer, SHA512_DIGESTLEN);
	sha->len = SHA512_BLOCKLEN;
	
	sha512_update(sha, md, SHA512_DIGESTLEN);
	sha512_final(sha, md); // md = D(outer || D(inner || msg))
	
	// reset sha to inner state -> reset hmac
    my_cuda_memcpy_uint64(sha->h, hmac->h_inner, SHA512_DIGESTLEN);
	sha->len = SHA512_BLOCKLEN;
}

__device__ PBKDF2_SHA512_DEF void pbkdf2_sha512(HMAC_SHA512_CTX *hmac,
    const uint8_t *key, uint32_t keylen, const uint8_t *salt, uint32_t saltlen, uint32_t rounds,
    uint8_t *dk, uint32_t dklen)
{
	uint32_t hlen = SHA512_DIGESTLEN;
	uint32_t l = dklen / hlen + ((dklen % hlen) ? 1 : 0);
	uint32_t r = dklen - (l - 1) * hlen;
	
	hmac_sha512_init(hmac, key, keylen);
	
	uint8_t *U = hmac->buf;
	uint8_t *T = dk;
	uint8_t count[4];
	
	uint32_t i, j, k;
	uint32_t len = hlen;
	for (i = 1; i <= l; i++)
	{
		if (i == l) { len = r; }
		count[0] = (i >> 24) & 0xFF;
		count[1] = (i >> 16) & 0xFF;
		count[2] = (i >>  8) & 0xFF;
		count[3] = (i) & 0xFF;
		hmac_sha512_update(hmac, salt, saltlen);
		hmac_sha512_update(hmac, count, 4);
		hmac_sha512_final(hmac, U);
        my_cuda_memcpy_unsigned_char(T, U, len);
		for (j = 1; j < rounds; j++)
		{
			hmac_sha512_update(hmac, U, hlen);
			hmac_sha512_final(hmac, U);
			for (k = 0; k < len; k++)
			{
				T[k] ^= U[k];
			}
		}
		T += len;
	}	
}

__device__ void compute_sha(const uint8_t *msg, uint32_t mlen)
{
	uint8_t md[SHA512_DIGESTLEN] = {0};  // Initialize to zero
    SHA512_CTX sha;
    sha512_init(&sha);
    sha512_update(&sha, msg, mlen);
    sha512_final(&sha, md);
    printf("SHA-512: ");
    print_as_hex(md, sizeof md);
}

__device__ void compute_hmac(const uint8_t *key, uint32_t klen, const uint8_t *msg, uint32_t mlen)
{
	uint8_t md[SHA512_DIGESTLEN];
	HMAC_SHA512_CTX hmac;
	hmac_sha512_init(&hmac, key, klen);
	hmac_sha512_update(&hmac, msg, mlen);
	hmac_sha512_final(&hmac, md);
    printf("HMAC: ");
	print_as_hex(md, sizeof md);
}

__device__ void compute_pbkdf2(
    const uint8_t *key,
    uint32_t klen,
    const uint8_t *salt,
    uint32_t slen,
    uint32_t rounds,
    uint32_t dklen,
    unsigned char *derived_key
    )
{
    uint8_t *dk = (uint8_t*) malloc(dklen);
	HMAC_SHA512_CTX pbkdf_hmac;
	pbkdf2_sha512(&pbkdf_hmac, key, klen, salt, slen, rounds, dk, dklen);
	printf("PBKDF2-SHA-512: ");
	print_as_hex(dk, dklen);
    my_cuda_memcpy_unsigned_char(derived_key, dk, dklen);
	free(dk);
}

__device__ unsigned char* Bip39SeedGenerator() {
    // Convert the mnemonic and passphrase to byte arrays (or use them as-is if you can)
    uint8_t *m_mnemonic = (unsigned char *)"sell stereo useless course suffer tribe jazz monster fresh excess wire again father film sudden pelican always room attack rubber pelican trash alone cancel";
    uint8_t *salt = (unsigned char *)"mnemonicTESTPHRASE";
    //unsigned char derived_key[64];  // This will hold the generated seed
    unsigned char* derived_key = (unsigned char*) malloc(64 * sizeof(unsigned char));
    // Initialize derived_key to zeros
    for (int i = 0; i < 64; ++i) {
        derived_key[i] = 0;
    }

    // compute_sha((uint8_t *) m_mnemonic, my_strlen((const char*) m_mnemonic));

    /*compute_hmac(
        (uint8_t *) m_mnemonic, 
        my_strlen((const char*) m_mnemonic), 
        (uint8_t *) salt, 
        my_strlen((const char*) salt)
        );*/

    // Call pbkdf2_hmac to perform the key derivation
    compute_pbkdf2(
        (uint8_t *) m_mnemonic, 
        my_strlen((const char*) m_mnemonic), 
        (uint8_t *) salt, 
        my_strlen((const char*) salt),
	    2048, 
        64,
        derived_key
        );
    // printf("Cuda derived_key: ");
    // print_as_hex(derived_key, 64);    
    return derived_key;
}
// -- Bip39SeedGenerator --

__device__ int string_compare(const char* str1, const char* str2, int length) {
    for (int i = 0; i < length; i++) {
        if (str1[i] != str2[i]) {
            return 0;
        }
    }
    return 1;
}

__global__ void my_kernel(
    char* mnemonic,
    char* computed_addresses,
    char* passphrases,
    char* target_addresses,
    unsigned char target_addresses_count,
    char* result
    ) 
    {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int LINE_LENGTH_ADDR = 45;
    int LINE_LENGTH_PASS = 10;
    int passphrase_idx = idx / 10;
    // Define the test_passphrase of length 10
    char test_passphrase[19]; //len('mnemonic') + len(passphrase)
    if (idx == 2309 && idx < 2310) {

        // print mnemonic
        printf("Cuda   salt: ");
        for (int i = 0; i < 156; i++) { // DOTO: Define your to mnemonic length
            printf("%c", mnemonic[i]);
        }
        printf("\n");
        
        // +++ Bip39SeedGenerator +++
        // add the "mnemonoic" text before the passphrase
        char mnemonic_word[9] = "mnemonic"; 
        for (int i = 0; i < 9; i++) {
            test_passphrase[i] = mnemonic_word[i];
        }
        // Fill the test_passphrase with the current passphrase
        for (int i = 0; i < LINE_LENGTH_PASS; i++) {
            test_passphrase[9+i] = passphrases[passphrase_idx * LINE_LENGTH_PASS + i];
        }        

        // print the test_passphrase
        printf("test_passphrase: ");
        for (int i = 0; i < LINE_LENGTH_PASS+9; i++) {
            printf("%c", test_passphrase[i]);
        }
        printf("\n");
        unsigned char* bip39seed = Bip39SeedGenerator();
        // print the bip39seed
        printf("bip39seed: ");
        for (int i = 0; i < 64; i++) {
            printf("%02X", bip39seed[i]);
        }
        printf("\n");
        // --- Bip39SeedGenerator ---


        for (int target_idx = 0; target_idx < target_addresses_count; target_idx++) {
            // print string_compare(&computed_addresses[idx * LINE_LENGTH_ADDR]
            printf("&computed_addresses[idx * LINE_LENGTH_ADDR]: ");
            for (int i = 0; i < LINE_LENGTH_ADDR; i++) {
                printf("%c", computed_addresses[idx * LINE_LENGTH_ADDR + i]);
            }
            printf("\n");
            // Compare the computed address with the target address
            if (string_compare(&computed_addresses[idx * LINE_LENGTH_ADDR], &target_addresses[target_idx * LINE_LENGTH_ADDR], LINE_LENGTH_ADDR)) {
                // Print that match was found for idx, phrase, and target
                printf("Match found for idx %d, phrase %d, and target %d\n", idx, passphrase_idx, target_idx);
                for (int i = 0; i < LINE_LENGTH_PASS; i++) {
                    result[i] = passphrases[passphrase_idx * LINE_LENGTH_PASS + i];
                }
            }
        }
    }
}

/* ### Datatypes ###
Length (bytes) NumPy type	CUDA type
1 np.int8	    signed char 2**7-1 == 127
2 np.int16	    short 2**15-1 == 32767
4 np.int32	    int 2**31-1 == 2147483647
8 np.int64	    long long 2**63-1 == 9223372036854775807
1 np.uint8	    unsigned char 2**8-1 == 255
2 np.uint16	    unsigned short 2**16-1 == 65535
4 np.uint32     unsigned int 2**32-1 == 4294967295
8 np.uint64     unsigned long long 2**64-1 == 18446744073709551615
4 np.float32	float 2**32-1 == 4294967295
8 np.float64    double 2**64-1 == 18446744073709551615
*/