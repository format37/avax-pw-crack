
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    *c = *a + *b;
}

int main()
{
    int a = 5, b = 7, c = 0;
    int *d_a, *d_b, *d_c;

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    // Copy inputs to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    addKernel<<<1, 1>>>(d_c, d_a, d_b);

    // Copy result back to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("Result: %d + %d = %d\n", a, b, c);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}