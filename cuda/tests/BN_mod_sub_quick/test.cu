#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "hip/hip_runtime.h"
#include ""
#include "bignum.h"

// Test case structure
struct sub_quick_test_case {
    const char* a_hex;    // First operand
    const char* b_hex;    // Second operand
    const char* n_hex;    // Modulus
};

// Convert hex string to BIGNUM_CUDA
__device__ void hex2bn(BIGNUM_CUDA* bn, const char* hex) {
    // Skip "0x" prefix if present
    if (hex[0] == '0' && (hex[1] == 'x' || hex[1] == 'X')) {
        hex += 2;
    }

    init_zero(bn);
    int len = 0;
    while (hex[len] != '\0') len++;
    
    int word_index = 0;
    BN_ULONG current_word = 0;
    int bits_in_current_word = 0;
    
    // Process hex string from right to left
    for (int i = len - 1; i >= 0; i--) {
        char c = hex[i];
        uint8_t value;
        
        if (c >= '0' && c <= '9') {
            value = c - '0';
        } else if (c >= 'a' && c <= 'f') {
            value = c - 'a' + 10;
        } else if (c >= 'A' && c <= 'F') {
            value = c - 'A' + 10;
        } else {
            continue;  // Skip invalid characters
        }

        current_word |= ((BN_ULONG)value << bits_in_current_word);
        bits_in_current_word += 4;

        if (bits_in_current_word >= BN_ULONG_NUM_BITS) {
            bn->d[word_index++] = current_word;
            current_word = 0;
            bits_in_current_word = 0;
        }
    }

    // Handle any remaining bits
    if (bits_in_current_word > 0) {
        bn->d[word_index++] = current_word;
    }

    bn->top = find_top_cuda(bn);
    bn->neg = 0;
}

// Print a BIGNUM in hex format with a label
__device__ void print_bn_hex(const char* label, const BIGNUM_CUDA* bn) {
    printf("%s: ", label);
    
    if (bn->top == 0 || (bn->top == 1 && bn->d[0] == 0)) {
        printf("0\n");
        return;
    }

    #ifdef BN_128
        // Handle 128-bit words
        for (int i = bn->top - 1; i >= 0; i--) {
            unsigned __int128 word = bn->d[i];
            uint64_t high = (uint64_t)(word >> 64);
            uint64_t low = (uint64_t)word;
            if (i == bn->top - 1) {
                if (high != 0) {
                    printf("%lX%016lX", high, low);
                } else {
                    printf("%lX", low);
                }
            } else {
                printf("%016lX%016lX", high, low);
            }
        }
    #else
        // Handle 64-bit words
        for (int i = bn->top - 1; i >= 0; i--) {
            if (i == bn->top - 1) {
                printf("%lX", bn->d[i]);
            } else {
                printf("%016lX", bn->d[i]);
            }
        }
    #endif
    printf("\n");
}

// Run a single test case
__device__ void run_sub_quick_test(const struct sub_quick_test_case* test) {
    BIGNUM_CUDA a, b, n, r;
    
    // Initialize numbers
    hex2bn(&a, test->a_hex);
    hex2bn(&b, test->b_hex);
    hex2bn(&n, test->n_hex);
    init_zero(&r);

    // Print input values
    printf("\nTest inputs:\n");
    print_bn_hex("a", &a);
    print_bn_hex("b", &b);
    print_bn_hex("n", &n);

    // Print bit lengths for debugging
    printf("\nBit lengths:\n");
    printf("a bits: %d\n", bn_bit_length(&a));
    printf("b bits: %d\n", bn_bit_length(&b));
    printf("n bits: %d\n", bn_bit_length(&n));

    // Perform modular subtraction
    if (!bn_mod_sub_quick(&r, &a, &b, &n)) {
        printf("Modular subtraction failed\n");
        return;
    }

    printf("\nResult:\n");
    print_bn_hex("r (final result)", &r);
}

__global__ void test_kernel() {
    // Define test cases
    struct sub_quick_test_case test_cases[] = {
        // Test Case 1: OpenSSL example (128-bit numbers)
        {
            "E10925726C3018DCB512F4EBF0A8835B",
            "EB772E27B51120720C3913490298D9A7",
            "FBB36E8A921F0B6E56E12B56CE3F0AD3"
        }
        // Add more test cases here as needed
    };

    // Run test cases
    size_t num_tests = sizeof(test_cases) / sizeof(test_cases[0]);
    for (size_t i = 0; i < num_tests; i++) {
        printf("\n=== Test case %zu ===\n", i + 1);
        run_sub_quick_test(&test_cases[i]);
    }
}

// Host function to launch the test
void run_bn_mod_sub_quick_test() {
    test_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}

int main(void) {
    // Run the CUDA test
    run_bn_mod_sub_quick_test();
    return 0;
}