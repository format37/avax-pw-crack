#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"

using namespace cuFIXNUM;

typedef warp_fixnum<64, u64_fixnum> fixnum;
typedef fixnum_array<fixnum> fixnum_array_t;

// This is our corrected big number multiplication function
__device__ void bn_mul(fixnum *result, fixnum *a, fixnum *b) {
    fixnum::mul_lo(*result, *a, *b);
}

// Kernel function to perform the multiplication
__global__ void mul_kernel(fixnum *result, fixnum *a, fixnum *b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < fixnum::SLOT_WIDTH) {
        bn_mul(&result[idx], &a[idx], &b[idx]);
    }
}

void initialize_number(uint8_t* num, int size, const char* hex_string) {
    memset(num, 0, size);
    int len = strlen(hex_string);
    for (int i = 0; i < len; i += 2) {
        int value;
        sscanf(hex_string + len - i - 2, "%2x", &value);
        num[i / 2] = value;
    }
}

void print_number(const char* label, const uint8_t* num, int size) {
    printf("%s", label);
    for (int i = size - 1; i >= 0; --i) {
        printf("%02x", num[i]);
    }
    printf("\n");
}

int main() {
    uint8_t num1[64];
    uint8_t num2[64];

    // Initialize num1 to a large 512-bit number
    initialize_number(num1, sizeof(num1), "ffffffffffffffffffffffffffffffffffffffffffffffffffffffffffffffff");

    // Initialize num2 to another large number
    initialize_number(num2, sizeof(num2), "aaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa");

    fixnum_array_t *a = fixnum_array_t::create(num1, sizeof(num1), sizeof(num1));
    fixnum_array_t *b = fixnum_array_t::create(num2, sizeof(num2), sizeof(num2));
    fixnum_array_t *result = fixnum_array_t::create(1);

    // Launch the kernel
    mul_kernel<<<1, fixnum::SLOT_WIDTH>>>((fixnum*)result->get_ptr(), (fixnum*)a->get_ptr(), (fixnum*)b->get_ptr());

    // Wait for GPU to finish
    hipDeviceSynchronize();

    uint8_t output[64];
    int nelts;
    result->retrieve_all(output, sizeof(output), &nelts);

    print_number("a: ", num1, sizeof(num1));
    print_number("b: ", num2, sizeof(num2));
    print_number("Result: ", output, sizeof(output));

    delete a;
    delete b;
    delete result;

    return 0;
}