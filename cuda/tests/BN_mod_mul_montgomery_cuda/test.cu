#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bignum.h"

// Test case structure 
__device__ struct mont_test_case {
    const char* a_hex;   
    const char* b_hex;    
    const char* n_hex;    
};

// Convert hex string to BIGNUM_CUDA
__device__ void hex2bn_cuda(BIGNUM_CUDA *bn, const char *hex) {
    init_zero(bn);
    int len = 0;
    while(hex[len] != '\0') len++;
    
    // Process hex string in chunks of 16 chars (64 bits)
    int chunks = (len + 15) / 16;
    for(int i = 0; i < chunks && i < MAX_BIGNUM_SIZE; i++) {
        BN_ULONG val = 0;
        int start = len - (i + 1) * 16;
        if(start < 0) start = 0;
        int chunk_size = len - i * 16 - start;
        
        for(int j = 0; j < chunk_size; j++) {
            char c = hex[start + j];
            int digit;
            if(c >= '0' && c <= '9') digit = c - '0';
            else if(c >= 'A' && c <= 'F') digit = c - 'A' + 10;
            else if(c >= 'a' && c <= 'f') digit = c - 'a' + 10;
            else continue;
            
            val = (val << 4) | digit;
        }
        bn->d[i] = val;
        if(val != 0) bn->top = i + 1;
    }
    if(bn->top == 0) bn->top = 1;
}

__device__ struct mont_test_case test_cases[] = {
    // Test 1: Small numbers (32-bit)
    {
        "11111111",  // a
        "22222222",  // b
        "FFFFFFFF"   // n
    },
    
    // Test 2: 64-bit numbers
    {
        "FFFFFFFFFFFFFFFF",
        "FFFFFFFFFFFFFFFF", 
        "FFFFFFFFFFFFFFFD"
    },
    
    // Test 3: 128-bit numbers
    {
        "FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFF",
        "FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFF",
        "FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFD"
    },
    
    // Test 4: 384-bit numbers
    {
        "FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFFFF0000000000000000FFFFFFFF",
        "FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFF",
        "FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFFFF0000000000000000FFFFFFFD"
    }
};

// Test function
__device__ void run_mont_test(const struct mont_test_case *test) {
    // Create bignums
    BIGNUM_CUDA a, b, n, r;
    init_zero(&a);
    init_zero(&b);
    init_zero(&n); 
    init_zero(&r);
    
    // Convert test values
    hex2bn_cuda(&a, test->a_hex);
    hex2bn_cuda(&b, test->b_hex);
    hex2bn_cuda(&n, test->n_hex);
    
    // Create and initialize Montgomery context
    BN_MONT_CTX_CUDA *mont = BN_MONT_CTX_new_cuda();
    if(mont == NULL) {
        printf("Failed to create Montgomery context\n");
        return;
    }
    
    // Set up Montgomery context 
    if(!BN_MONT_CTX_set_cuda(mont, &n)) {
        printf("Failed to initialize Montgomery context\n");
        return;
    }
    
    printf("\nTest inputs:\n");
    bn_print_no_fuse("a: ", &a);
    bn_print_no_fuse("b: ", &b);
    bn_print_no_fuse("n: ", &n);

    // Convert to Montgomery form 
    BIGNUM_CUDA aRR, bRR;
    init_zero(&aRR);
    init_zero(&bRR);
    
    if(!BN_mod_mul_montgomery_cuda(&aRR, &a, &mont->RR, mont) ||
       !BN_mod_mul_montgomery_cuda(&bRR, &b, &mont->RR, mont)) {
        printf("Failed to convert to Montgomery form\n");
        return;
    }
    
    printf("\nMontgomery form (RR values):\n"); 
    bn_print_no_fuse("aRR: ", &aRR);
    bn_print_no_fuse("bRR: ", &bRR);

    // Perform Montgomery multiplication
    if(!BN_mod_mul_montgomery_cuda(&r, &aRR, &bRR, mont)) {
        printf("Montgomery multiplication failed\n");
        return; 
    }

    printf("\nResult:\n");
    bn_print_no_fuse("r (Montgomery form): ", &r);
    
    // Convert back from Montgomery form
    BIGNUM_CUDA final;
    init_zero(&final);
    BIGNUM_CUDA one; 
    init_one(&one);
    
    if(!BN_mod_mul_montgomery_cuda(&final, &r, &one, mont)) {
        printf("Failed to convert back from Montgomery form\n");
        return;
    }
    
    bn_print_no_fuse("r (final result): ", &final);
    
    printf("\nMontgomery Context:\n");
    bn_print_no_fuse("N (modulus): ", &mont->N);
    printf("N0: [%016llx, %016llx]\n", mont->n0[0], mont->n0[1]);
    bn_print_no_fuse("RR: ", &mont->RR);
    
    printf("\n");
    
    // Cleanup
    free(mont);
}

__global__ void test_montgomery() {
    for(int i = 0; i < sizeof(test_cases)/sizeof(test_cases[0]); i++) {
        printf("\n=== Test case %d ===\n", i + 1);
        run_mont_test(&test_cases[i]);
    }
}

int main() {
    test_montgomery<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}