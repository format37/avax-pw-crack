#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Include your CUDA header files here
#include "bignum.h"
#include "point.h"
#include "public_key.h"

#define MAX_LINE_LENGTH 1024
#define MAX_TEST_CASES 1000
#define HEX_STRING_LENGTH 65  // 64 characters for 256-bit number + null terminator

// Structure to hold a test case
typedef struct {
    char Px[HEX_STRING_LENGTH], Py[HEX_STRING_LENGTH], Qx[HEX_STRING_LENGTH], Qy[HEX_STRING_LENGTH];
    char ExpectedAddX[HEX_STRING_LENGTH], ExpectedAddY[HEX_STRING_LENGTH];
    char ExpectedDoubleX[HEX_STRING_LENGTH], ExpectedDoubleY[HEX_STRING_LENGTH];
} TestCase;

// Device function to calculate string length
__device__ int d_strlen(const char *str) {
    int len = 0;
    while (str[len] != '\0') {
        len++;
    }
    return len;
}

// Device function to copy n characters from src to dest
__device__ char* d_strncpy(char *dest, const char *src, int n) {
    int i;
    for (i = 0; i < n && src[i] != '\0'; i++) {
        dest[i] = src[i];
    }
    for (; i < n; i++) {
        dest[i] = '\0';
    }
    return dest;
}

// Device function to convert hex string to unsigned long long
__device__ unsigned long long d_strtoull(const char *str, char **endptr, int base) {
    unsigned long long result = 0;
    int i = 0;

    while (str[i] != '\0') {
        int digit;
        if (str[i] >= '0' && str[i] <= '9') {
            digit = str[i] - '0';
        } else if (str[i] >= 'a' && str[i] <= 'f') {
            digit = str[i] - 'a' + 10;
        } else if (str[i] >= 'A' && str[i] <= 'F') {
            digit = str[i] - 'A' + 10;
        } else {
            break;
        }
        
        if (digit >= base) {
            break;
        }
        
        result = result * base + digit;
        i++;
    }

    if (endptr) {
        *endptr = (char*)str + i;
    }

    return result;
}

// Function to initialize a BIGNUM from a hex string
__device__ void initBignumFromHex_err(BIGNUM *bn, const char *hex) {
    init_zero(bn);
    int len = d_strlen(hex);
    for (int i = 0; i < len; i += 16) {
        char chunk[17] = {0};
        int chunk_len = (len - i < 16) ? (len - i) : 16;
        d_strncpy(chunk, hex + len - i - chunk_len, chunk_len);
        BN_ULONG word = d_strtoull(chunk, NULL, 16);
        bn->d[bn->top++] = word;
    }
    bn->top = find_top(bn);
}

__device__ void initBignumFromHex(BIGNUM *bn, const char *hex) {
    init_zero(bn);
    int len = d_strlen(hex);
    int word_index = 0;

    for (int i = len; i > 0; i -= 16) {
        char chunk[17] = {0};
        int chunk_len = (i < 16) ? i : 16;
        d_strncpy(chunk, hex + i - chunk_len, chunk_len);
        BN_ULONG word = d_strtoull(chunk, NULL, 16);
        bn->d[word_index++] = word;
    }

    bn->top = find_top(bn);
}

// CUDA kernel to perform the tests
__global__ void testEllipticCurve(TestCase *cases, int numCases) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numCases) return;

    TestCase *tc = &cases[idx];
    EC_POINT_CUDA P, Q, resultAdd, resultDouble;

    // Initialize points P and Q
    initBignumFromHex(&P.x, tc->Px);
    initBignumFromHex(&P.y, tc->Py);
    initBignumFromHex(&Q.x, tc->Qx);
    initBignumFromHex(&Q.y, tc->Qy);

    // Perform point addition
    const BIGNUM CURVE_A_LOCAL = {0};
    const BIGNUM CURVE_P_LOCAL = {
        {
            0xFFFFFFFEFFFFFC2F,
            0xFFFFFFFFFFFFFFFF,
            0xFFFFFFFFFFFFFFFF,
            0xFFFFFFFFFFFFFFFF
        },
        CURVE_P_VALUES_MAX_SIZE,
        false
    };

    point_add(&resultAdd, &P, &Q, &CURVE_P_LOCAL, &CURVE_A_LOCAL);
    // Perform point doubling
    point_add(&resultDouble, &P, &P, &CURVE_P_LOCAL, &CURVE_A_LOCAL);
    // Initialize expected results
    BIGNUM expectedAddX, expectedAddY, expectedDoubleX, expectedDoubleY;
    initBignumFromHex(&expectedAddX, tc->ExpectedAddX);
    initBignumFromHex(&expectedAddY, tc->ExpectedAddY);
    initBignumFromHex(&expectedDoubleX, tc->ExpectedDoubleX);
    initBignumFromHex(&expectedDoubleY, tc->ExpectedDoubleY);

    // Compare results
    bool additionCorrect = (bn_cmp(&resultAdd.x, &expectedAddX) == 0) &&
                           (bn_cmp(&resultAdd.y, &expectedAddY) == 0);
    bool doublingCorrect = (bn_cmp(&resultDouble.x, &expectedDoubleX) == 0) &&
                           (bn_cmp(&resultDouble.y, &expectedDoubleY) == 0);

    // Print results
    printf("Test case %d: Addition %s, Doubling %s\n", idx,
           additionCorrect ? "PASS" : "FAIL",
           doublingCorrect ? "PASS" : "FAIL");
}

// Host function to read test cases from file
int readTestCases(const char *filename, TestCase *cases) {
    FILE *file = fopen(filename, "r");
    if (!file) {
        perror("Error opening file");
        return -1;
    }

    char line[MAX_LINE_LENGTH];
    int numCases = 0;

    while (fgets(line, sizeof(line), file) && numCases < MAX_TEST_CASES) {
        TestCase *tc = &cases[numCases];
        sscanf(line, "%64s %64s %64s %64s %64s %64s %64s %64s",
               tc->Px, tc->Py, tc->Qx, tc->Qy,
               tc->ExpectedAddX, tc->ExpectedAddY,
               tc->ExpectedDoubleX, tc->ExpectedDoubleY);
        numCases++;
    }

    fclose(file);
    return numCases;
}

int main() {
    TestCase *h_cases, *d_cases;
    int numCases;

    // Read test cases from file
    h_cases = (TestCase*)malloc(MAX_TEST_CASES * sizeof(TestCase));
    numCases = readTestCases("../../../point_add_cases_full.txt", h_cases);
    if (numCases < 0) {
        fprintf(stderr, "Failed to read test cases\n");
        return 1;
    }

    // Allocate memory on device
    hipMalloc(&d_cases, numCases * sizeof(TestCase));

    // Copy data to device
    hipMemcpy(d_cases, h_cases, numCases * sizeof(TestCase), hipMemcpyHostToDevice);

    // Launch kernel
    // int threadsPerBlock = 256;
    int threadsPerBlock = 1;
    int blocksPerGrid = (numCases + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    testEllipticCurve<<<blocksPerGrid, threadsPerBlock>>>(d_cases, numCases);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    printf("Done\n");

    // Free device memory
    hipFree(d_cases);
    
    // Free host memory
    free(h_cases);

    return 0;
}