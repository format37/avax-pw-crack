#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "bignum.h"
#include "montgomery.h"
#include "point.h"

__device__ void init_test_vectors(
    EC_GROUP_CUDA *group,         // Curve parameters
    EC_POINT_JACOBIAN *base_point,    // P in affine coordinates
    EC_POINT_JACOBIAN *r_point,       // R in projective coordinates 
    EC_POINT_JACOBIAN *s_point        // S in projective coordinates
) {
    // Initialize group parameters
    // Group: secp256k1
    // Field: FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F
    
    // Field prime
    init_zero(&group->field);
    group->field.d[3] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[2] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[1] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[0] = 0xFFFFFFFEFFFFFC2F;
    group->field.top = 4;
    group->field.neg = false;

    // a = 0
    init_zero(&group->a);
    group->a.top = 1;
    group->a.neg = false;

    // b = 7
    init_zero(&group->b); 
    group->b.d[0] = 0x0700001AB7;
    group->b.top = 1;
    group->b.neg = false;

    // Initialize curve order
    init_zero(&group->order);
    group->order.d[3] = 0xFFFFFFFFFFFFFFFF;
    group->order.d[2] = 0xFFFFFFFFFFFFFFFE;
    group->order.d[1] = 0xBAAEDCE6AF48A03B;
    group->order.d[0] = 0xBFD25E8CD0364141;
    group->order.top = 4;
    group->order.neg = false;
    
    // Initialize base point P (secp256k1 generator point)
    // x coordinate
    init_zero(&base_point->X);
    base_point->X.d[3] = 0x9981E643E9089F48;
    base_point->X.d[2] = 0x979F48C033FD129C;
    base_point->X.d[1] = 0x231E295329BC66DB;
    base_point->X.d[0] = 0xD7362E5A487E2097;
    base_point->X.top = 4;
    base_point->X.neg = false;
    // y coordinate
    init_zero(&base_point->Y);
    base_point->Y.d[3] = 0xCF3F851FD4A582D6;
    base_point->Y.d[2] = 0x70B6B59AAC19C136;
    base_point->Y.d[1] = 0x8DFC5D5D1F1DC64D;
    base_point->Y.d[0] = 0xB15EA6D2D3DBABE2;
    base_point->Y.top = 4;
    base_point->Y.neg = false;
    // z coordinate
    init_zero(&base_point->Z);
    base_point->Z.d[0] = 0x01000003D1;
    base_point->Z.top = 1;
    base_point->Z.neg = false;

    // Set S point
    // x coordinate
    init_zero(&s_point->X);
    s_point->X.d[3] = 0x6905E1FC2278EFBB;
    s_point->X.d[2] = 0x2636020B60463145;
    s_point->X.d[1] = 0x289DDE613CBA2A22;
    s_point->X.d[0] = 0xB99C9635D9FDF005;
    s_point->X.top = 4;
    s_point->X.neg = false;
    // y coordinate
    init_zero(&s_point->Y);
    s_point->Y.d[3] = 0x30BA5EDD40D38D1F;
    s_point->Y.d[2] = 0x5B24C02BF347D852;
    s_point->Y.d[1] = 0xB7E5BD29A318FE73;
    s_point->Y.d[0] = 0x230398906311A839;
    s_point->Y.top = 4;
    s_point->Y.neg = false;
    // z coordinate
    init_zero(&s_point->Z);
    s_point->Z.d[3] = 0xDFA30ABA85E5DF8C;
    s_point->Z.d[2] = 0x7C231D3677E79480;
    s_point->Z.d[1] = 0x43812611FE566CF4;
    s_point->Z.d[0] = 0xF1FD34A59A26A44F;
    s_point->Z.top = 4;
    s_point->Z.neg = false;
    
    // Initialize R point
    // x coordinate
    init_zero(&r_point->X);
    r_point->X.d[3] = 0xA22EFED5BB4CEB00;
    r_point->X.d[2] = 0x243A15E5A0026879;
    r_point->X.d[1] = 0xDB96B2D2AD4BD252;
    r_point->X.d[0] = 0xBF22A6C2C5871863;
    r_point->X.top = 4;
    r_point->X.neg = false;

    // y coordinate
    init_zero(&r_point->Y);
    r_point->Y.d[3] = 0x946A5EDAF9E2D7E1;
    r_point->Y.d[2] = 0x2AD7EA0B5F5C1227;
    r_point->Y.d[1] = 0xAE990A3121368017;
    r_point->Y.d[0] = 0x13DA23E0DB979EF9;
    r_point->Y.top = 4;
    r_point->Y.neg = false;

    // z coordinate
    init_zero(&r_point->Z);
    r_point->Z.d[3] = 0xB7B83ADE508E1534;
    r_point->Z.d[2] = 0x7CB092DEBA1E3198;
    r_point->Z.d[1] = 0xF51B942FA0773EB7;
    r_point->Z.d[0] = 0xAFDFECF39D69ABF9;
    r_point->Z.top = 4;
    r_point->Z.neg = false;    
}

__device__ void init_test_vectors_x(
    EC_GROUP_CUDA *group,         // Curve parameters
    EC_POINT_JACOBIAN *base_point,    // P in affine coordinates
    EC_POINT_JACOBIAN *r_point,       // R in projective coordinates 
    EC_POINT_JACOBIAN *s_point        // S in projective coordinates
) {
    // Initialize group parameters
    // Group: secp256k1
    // Field: FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEFFFFFC2F
    // A: 0
    // B: 0700001AB7
    // Order: FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141
    // Field prime
    init_zero(&group->field);
    group->field.d[3] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[2] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[1] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[0] = 0xFFFFFFFEFFFFFC2F;
    group->field.top = 4;
    group->field.neg = false;

    // a = 0
    init_zero(&group->a);
    group->a.top = 1;
    group->a.neg = false;

    // b = 7
    init_zero(&group->b); 
    group->b.d[0] = 0x0700001AB7;
    group->b.top = 1;
    group->b.neg = false;

    // Initialize curve order
    init_zero(&group->order);
    group->order.d[3] = 0xFFFFFFFFFFFFFFFF;
    group->order.d[2] = 0xFFFFFFFFFFFFFFFE;
    group->order.d[1] = 0xBAAEDCE6AF48A03B;
    group->order.d[0] = 0xBFD25E8CD0364141;
    group->order.top = 4;
    group->order.neg = false;
    
    // Initialize base point P (secp256k1 generator point)
    // [1] Initial point: p (generator): 0479BE667EF9DCBBAC55A06295CE870B07029BFCDB2DCE28D959F2815B16F81798483ADA7726A3C4655DA4FBFC0E1108A8FD17B448A68554199C47D08FFB10D4B8
    // X: 9981E643E9089F48979F48C033FD129C231E295329BC66DBD7362E5A487E2097
    // Y: CF3F851FD4A582D670B6B59AAC19C1368DFC5D5D1F1DC64DB15EA6D2D3DBABE2
    // Z: 01000003D1
    // Z_is_one: 1
    // EC_POINT_CUDA base_point;


    // bn_print_no_fuse("&base_point.X:", &base_point->X);
    // return;
    // x coordinate
    init_zero(&base_point->X);
    // // 9981E643E9089F48 979F48C033FD129C 231E295329BC66DB D7362E5A487E2097
    // base_point->X.d[3] = 0x9981E643E9089F48;
    // base_point->X.d[2] = 0x979F48C033FD129C;
    // base_point->X.d[1] = 0x231E295329BC66DB;
    // base_point->X.d[0] = 0xD7362E5A487E2097;
    // 9981E643E9089F48979F48C033FD129C231E295329BC66DBD7362E5A487E2097
    base_point->X.d[3] = 0x9981E643E9089F48;
    base_point->X.d[2] = 0x979F48C033FD129C;
    base_point->X.d[1] = 0x231E295329BC66DB;
    base_point->X.d[0] = 0xD7362E5A487E2097;
    base_point->X.top = 4;
    base_point->X.neg = false;

    // y coordinate
    init_zero(&base_point->Y);
    // // CF3F851FD4A582D6 70B6B59AAC19C136 8DFC5D5D1F1DC64D B15EA6D2D3DBABE2
    // base_point->Y.d[3] = 0xCF3F851FD4A582D6;
    // base_point->Y.d[2] = 0x70B6B59AAC19C136;
    // base_point->Y.d[1] = 0x8DFC5D5D1F1DC64D;
    // base_point->Y.d[0] = 0xB15EA6D2D3DBABE2;
    // CF3F851FD4A582D670B6B59AAC19C1368DFC5D5D1F1DC64DB15EA6D2D3DBABE2
    base_point->Y.d[3] = 0xCF3F851FD4A582D6;
    base_point->Y.d[2] = 0x70B6B59AAC19C136;
    base_point->Y.d[1] = 0x8DFC5D5D1F1DC64D;
    base_point->Y.d[0] = 0xB15EA6D2D3DBABE2;
    base_point->Y.top = 4;
    base_point->Y.neg = false;

    // z coordinate
    init_zero(&base_point->Z);
    // 00000001000003D1
    base_point->Z.d[0] = 0x00000001000003D1;
    base_point->Z.top = 1;
    base_point->Z.neg = false;

    // Copy S point from base_point
    // copy_jacobian_point(s_point, base_point);

    // Set S point accroding to values
    // x coordinate
    init_zero(&s_point->X);
    // 5D9C7BE194E8397AEA4C6964322235A3929D1128F00DFF18754526BC130E97B0
    s_point->X.d[3] = 0x5D9C7BE194E8397A;
    s_point->X.d[2] = 0xEA4C6964322235A3;
    s_point->X.d[1] = 0x929D1128F00DFF18;
    s_point->X.d[0] = 0x754526BC130E97B0;
    s_point->X.top = 4;
    s_point->X.neg = false;

    // y coordinate
    init_zero(&s_point->Y);
    // 9EB7840D46A6404D75DC64D82655BE6D1508B8C1F20C625E25C01D758F5A0271
    s_point->Y.d[3] = 0x9EB7840D46A6404D;
    s_point->Y.d[2] = 0x75DC64D82655BE6D;
    s_point->Y.d[1] = 0x1508B8C1F20C625E;
    s_point->Y.d[0] = 0x25C01D758F5A0271;
    s_point->Y.top = 4;
    s_point->Y.neg = false;

    // z coordinate
    init_zero(&s_point->Z);
    // E128853ACB0D914C79228AE1942C5D20E75E52284953EDE956FD766E7EB378DA
    s_point->Z.d[3] = 0xE128853ACB0D914C;
    s_point->Z.d[2] = 0x79228AE1942C5D20;
    s_point->Z.d[1] = 0xE75E52284953EDE9;
    s_point->Z.d[0] = 0x56FD766E7EB378DA;
    s_point->Z.top = 4;
    s_point->Z.neg = false;

    // Initialize R point
    // x coordinate
    init_zero(&r_point->X);
    // // 7C75DD9524177D59 3C03889B8DCD9B1C B05FB7D2A3DA7FE8 BA9F29B104E7DB13
    // r_point->X.d[3] = 0x7C75DD9524177D59;
    // r_point->X.d[2] = 0x3C03889B8DCD9B1C;
    // r_point->X.d[1] = 0xB05FB7D2A3DA7FE8;
    // r_point->X.d[0] = 0xBA9F29B104E7DB13;
    // 8972D7419759E11A C13A11538F557C3A 5760A16981ECFE3E BCBB84BC41A263E0
    r_point->X.d[3] = 0x8972D7419759E11A;
    r_point->X.d[2] = 0xC13A11538F557C3A;
    r_point->X.d[1] = 0x5760A16981ECFE3E;
    r_point->X.d[0] = 0xBCBB84BC41A263E0;
    r_point->X.top = 4;
    r_point->X.neg = false;

    // y coordinate
    init_zero(&r_point->Y);
    // // 55DEBB381F4AD034 CC27CB48A46449AA A87D43FDB563384B 1CD20838E6FDDC9F
    // r_point->Y.d[3] = 0x55DEBB381F4AD034;
    // r_point->Y.d[2] = 0xCC27CB48A46449AA;
    // r_point->Y.d[1] = 0xA87D43FDB563384B;
    // r_point->Y.d[0] = 0x1CD20838E6FDDC9F;
    // 946A5EDAF9E2D7E1 2AD7EA0B5F5C1227 AE990A3121368017 13DA23E0DB979EF9
    r_point->Y.d[3] = 0x946A5EDAF9E2D7E1;
    r_point->Y.d[2] = 0x2AD7EA0B5F5C1227;
    r_point->Y.d[1] = 0xAE990A3121368017;
    r_point->Y.d[0] = 0x13DA23E0DB979EF9;
    r_point->Y.top = 4;
    r_point->Y.neg = false;

    // z coordinate
    init_zero(&r_point->Z);
    // // 9E7F0A3FA94B05AC E16D6B355833826D 1BF8BABA3E3B8C9B 62BD4DA6A7B75B95
    // r_point->Z.d[3] = 0x9E7F0A3FA94B05AC;
    // r_point->Z.d[2] = 0xE16D6B355833826D;
    // r_point->Z.d[1] = 0x1BF8BABA3E3B8C9B;
    // r_point->Z.d[0] = 0x62BD4DA6A7B75B95;
    // 9028989D4A18DA0C0BD63FE04C06B8350379EF051DFFD3D649119832D1416931
    r_point->Z.d[3] = 0x9028989D4A18DA0C;
    r_point->Z.d[2] = 0x0BD63FE04C06B835;
    r_point->Z.d[1] = 0x0379EF051DFFD3D6;
    r_point->Z.d[0] = 0x49119832D1416931;
    r_point->Z.top = 4;
    r_point->Z.neg = false;    

    // Convert base point to Jacobian coordinates (s_point)
    // affine_to_jacobian(&base_point, s_point);

    // Compute r_point = 2P using point doubling in Jacobian coordinates
    // jacobian_point_double(r_point, s_point, &group->field, &group->a);
}

__device__ void print_point(const char* label, const EC_POINT_CUDA *point) {
    printf("%s:\n", label);
    bn_print_no_fuse("  x: ", &point->x);
    bn_print_no_fuse("  y: ", &point->y);
}

// In your main kernel function
__global__ void test_ladder_step() {
    printf("Starting ladder step test...\n");

    // Initialize group and points
    EC_GROUP_CUDA group;
    EC_POINT_JACOBIAN p_point, r_point, s_point;

    // Initialize test vectors
    init_test_vectors(&group, &p_point, &r_point, &s_point);

    printf("Test vectors initialized.\n");
    bn_print_no_fuse("Field prime: ", &group.field);
    bn_print_no_fuse("Base point a:", &group.a);
    bn_print_no_fuse("Base point b:", &group.b);
    bn_print_no_fuse("Curve order: ", &group.order);    
    print_jacobian_point("P point", &p_point);
    print_jacobian_point("R point", &r_point);
    print_jacobian_point("S point", &s_point);
    // return; // TODO: Remove this line

    // Perform ladder step
    int result = ec_point_ladder_step(&group, &r_point, &s_point, &p_point);

    if (result == 0) {
        printf("Ladder step operation failed!\n");
        return;
    }

    // Print results
    print_jacobian_point("Result R point", &r_point);
    print_jacobian_point("Result S point", &s_point);
    print_jacobian_point("Result P point", &p_point);
    printf("Ladder step test complete.\n");
}

int main() {
    // Set stack size
    size_t stackSize = 64 * 1024;  // 64KB
    hipDeviceSetLimit(hipLimitStackSize, stackSize);

    // Launch kernel
    test_ladder_step<<<1,1>>>();
    hipDeviceSynchronize();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }

    return 0;
}