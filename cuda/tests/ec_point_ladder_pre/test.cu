#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"
#include "montgomery.h"
#include "point.h"

__device__ void init_test_vectors(
    EC_GROUP_CUDA *group,         // Curve parameters 
    EC_POINT_JACOBIAN *p_point        // base point
) {
    // Initialize secp256k1 curve parameters
    init_zero(&group->field);
    group->field.d[3] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[2] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[1] = 0xFFFFFFFFFFFFFFFF;
    group->field.d[0] = 0xFFFFFFFEFFFFFC2F;
    group->field.top = 4;
    group->field.neg = false;

    // a = 0
    init_zero(&group->a);
    group->a.top = 1;
    group->a.neg = false;

    // b = 7 
    init_zero(&group->b);
    group->b.d[0] = 7;
    group->b.top = 1;
    group->b.neg = false;

    // Initialize curve order
    init_zero(&group->order);
    group->order.d[3] = 0xFFFFFFFFFFFFFFFF;
    group->order.d[2] = 0xFFFFFFFFFFFFFFFE;
    group->order.d[1] = 0xBAAEDCE6AF48A03B;
    group->order.d[0] = 0xBFD25E8CD0364141;
    group->order.top = 4;
    group->order.neg = false;
    
    // Initialize base point P (secp256k1 generator) in affine coordinates
    init_zero(&p_point->X);
    p_point->X.d[3] = 0x79BE667EF9DCBBAC;
    p_point->X.d[2] = 0x55A06295CE870B07;
    p_point->X.d[1] = 0x029BFCDB2DCE28D9;  
    p_point->X.d[0] = 0x59F2815B16F81798;
    p_point->X.top = 4;
    p_point->X.neg = false;

    init_zero(&p_point->Y); 
    p_point->Y.d[3] = 0x483ADA7726A3C465;
    p_point->Y.d[2] = 0x5DA4FBFC0E1108A8;
    p_point->Y.d[1] = 0xFD17B448A6855419; 
    p_point->Y.d[0] = 0x9C47D08FFB10D4B8;
    p_point->Y.top = 4;
    p_point->Y.neg = false;

    // Initialize Z coordinate to 1
    init_zero(&p_point->Z);
    p_point->Z.d[0] = 1;
    p_point->Z.top = 1;
    p_point->Z.neg = false;
}

__global__ void test_ladder_pre() {
    printf("Test EC point ladder pre...\n");

    // Initialize test vectors
    EC_GROUP_CUDA group;
    EC_POINT_JACOBIAN p, r, s;

    // Initialize points
    init_jacobian_point(&r);
    init_jacobian_point(&s);

    // Setup test vectors
    init_test_vectors(&group, &p);

    printf("\nInitial state:\n"); 
    print_jacobian_point("P (base point)", &p);
    print_jacobian_point("R", &r);
    print_jacobian_point("S", &s);

    // Perform ladder pre step
    if (!ossl_ec_GFp_simple_ladder_pre(&group, &r, &s, &p)) {
        printf("Ladder pre operation failed!\n");
        return;
    }

    printf("\nAfter ladder pre:\n");
    print_jacobian_point("R", &r);
    print_jacobian_point("S", &s);
    print_jacobian_point("P (base point)", &p);
}

int main() {
    // Launch kernel 
    test_ladder_pre<<<1,1>>>();
    hipDeviceSynchronize();

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }

    return 0;
}