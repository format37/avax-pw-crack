#include <hip/hip_runtime.h>
#include <stdio.h>

#define MAX_PASSPHRASE_LENGTH 8

unsigned long long get_variant_id(const char* s) {
    const char* alphabet = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789 !@#$%^&*()-_=+[]{};:'\",.<>?/\\|~";
    int base = 94;
    int length = (int)strlen(s);

    unsigned long long offset = 0;
    unsigned long long value = 0;

    // Compute offsets for all shorter lengths
    for (int i = 1; i < length; i++) {
        unsigned long long count = 1;
        for (int j = 0; j < i; j++) {
            count *= base;
        }
        offset += count;
    }

    // Convert the current string to a base-94 number
    for (int i = 0; i < length; i++) {
        const char* pos = strchr(alphabet, s[i]);
        if (pos != NULL) {
            int index = (int)(pos - alphabet);
            value = value * base + index;
        }
    }

    // Final ID is offset + computed base-94 value
    return offset + value;
}

__device__ void find_letter_variant(unsigned long long variant_id, char* result) {
    const char alphabet[] = "ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz0123456789 !@#$%^&*()-_=+[]{};:'\",.<>?/\\|~";
    const int base = 94;

    // Clear result
    for (int i = 0; i < MAX_PASSPHRASE_LENGTH; i++) {
        result[i] = '\0';
    }

    // Determine length by finding which range variant_id falls into
    int length = 1;
    unsigned long long total_count = 0;
    while (true) {
        unsigned long long count = 1;
        for (int j = 0; j < length; j++) {
            count *= base;
        }
        
        if (variant_id < total_count + count) {
            variant_id -= total_count;
            break;
        } else {
            total_count += count;
            length++;
            if (length > MAX_PASSPHRASE_LENGTH) {
                // Safety check: if too large, clamp
                length = MAX_PASSPHRASE_LENGTH;
                break;
            }
        }
    }

    // Decode variant_id as a base-94 number into 'length' characters
    unsigned long long temp = variant_id;
    for (int i = length - 1; i >= 0; i--) {
        int remainder = (int)(temp % base);
        temp /= base;
        result[i] = alphabet[remainder];
        if (i == 0) break; // to avoid infinite loop since i is unsigned
    }
}

__global__ void generate_variants_kernel(unsigned long long start_id, unsigned long long end_id) {
    char result[MAX_PASSPHRASE_LENGTH];
    unsigned long long current_id = start_id;
    
    while (current_id <= end_id) {
        find_letter_variant(current_id, result);
        printf("%s\n", result);
        current_id++;
    }
}

int main() {
    const char* start_phrase = "A";
    const char* end_phrase = "CC";
    
    // Convert strings to variant IDs on host
    unsigned long long start_id = get_variant_id(start_phrase);
    unsigned long long end_id = get_variant_id(end_phrase);
    
    printf("Starting variant generation from ID %llu to %llu\n", start_id, end_id);
    
    // Launch kernel with 1 block and 1 thread for testing
    generate_variants_kernel<<<1, 1>>>(start_id, end_id);
    
    // Wait for kernel to finish
    hipDeviceSynchronize();
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    return 0;
}