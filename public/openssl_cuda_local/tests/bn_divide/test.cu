#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

// Define your BIGNUM structure based on your project definitions
#define MAX_BIGNUM_WORDS 20
#define BN_ULONG unsigned long long int
#define BN_ULONG_NUM_BITS (sizeof(BN_ULONG) * 8)

// Test kernel for bn_divide
__global__ void testKernel() {
    printf("++ testKernel for bn_divide ++\n");
    // Set the maximum number of test cases
    const int num_tests = 7;  // Updated number of tests
    // Initialize the word_num array
    int word_num[num_tests] = {1, 1, 1, 1, 1, 2, 3};

    BN_ULONG test_values_dividend[][MAX_BIGNUM_WORDS] = {
        {0x1}, // Test 1
        {0xF}, // Test 2
        {0xF}, // Test 3
        {0x17}, // Test 4 // 23 in decimal
        {0x1234567890ABCDEF}, // Test 5
        {0x1234567890ABCDEF, 0}, // Test 6
        {0x1234567890ABCDEF, 0, 0} // Test 7

    };

    BN_ULONG test_values_divisor[][MAX_BIGNUM_WORDS] = {
        {0x2},
        {0xF}, 
        {0x1},
        {0x5},
        {0x1},
        {0x1, 0}, 
        {0x1, 0, 0}
    };

    // Initialize 'dividend' and 'divisor' with test values for each test
    for (int test = 0; test < num_tests; ++test) {
        BIGNUM dividend, divisor, quotient, remainder;
        init_zero(&dividend, MAX_BIGNUM_WORDS);
        init_zero(&divisor, MAX_BIGNUM_WORDS);
        init_zero(&quotient, MAX_BIGNUM_WORDS);
        init_zero(&remainder, MAX_BIGNUM_WORDS);
        
        // Assign test values to 'dividend' and 'divisor', and initialize top accordingly
        for (int i = 0; i < MAX_BIGNUM_WORDS; ++i) {
            dividend.d[i] = test_values_dividend[test][i];
            divisor.d[i] = test_values_divisor[test][i];
        }
        
        dividend.top = word_num[test];
        divisor.top = word_num[test];

        // Test division
        bn_divide(&quotient, &remainder, &dividend, &divisor);

        // Print results
        printf("Test %d:\n", test + 1);
        bn_print("dividend : ", &dividend);
        bn_print("divisor  : ", &divisor);
        bn_print("quotient : ", &quotient);
        bn_print("remainder: ", &remainder);
    }
    printf("-- Finished testKernel for bn_divide --\n");
}

// Main function
int main() {
    printf("Starting bn_divide test\n");
    // Launch the kernel to run the test
    testKernel<<<1, 1>>>();

    // Check for any errors after running the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    return 0;
}