#include "hip/hip_runtime.h"
//bn_div_test.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

#define TEST_BIGNUM_WORDS 8

__device__ void reverse_order(BN_ULONG test_values_a[][TEST_BIGNUM_WORDS], BN_ULONG test_values_b[][TEST_BIGNUM_WORDS], size_t num_rows) {
    for (size_t i = 0; i < num_rows; i++) {
        for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
            BN_ULONG temp_a = test_values_a[i][j];
            test_values_a[i][j] = test_values_a[i][TEST_BIGNUM_WORDS - 1 - j];
            test_values_a[i][TEST_BIGNUM_WORDS - 1 - j] = temp_a;

            BN_ULONG temp_b = test_values_b[i][j];
            test_values_b[i][j] = test_values_b[i][TEST_BIGNUM_WORDS - 1 - j];
            test_values_b[i][TEST_BIGNUM_WORDS - 1 - j] = temp_b;
        }
    }
}

__global__ void testKernel() {
    /*BN_ULONG test_values_dividend[][MAX_BIGNUM_WORDS] = {
        {0xffffffffffffffff, 0xffffffffffffffe, 0xbaaedce6af48a03b, 0xbfd25e8cd0364141},
    };

    BN_ULONG test_values_divisor[][MAX_BIGNUM_WORDS] = {
        {0x1b2db4c027cdbaba, 0x70116675aa53aa8a, 0xad1c289591e564d3, 0xcaa5c571ffccab5a},
    };*/

    BN_ULONG test_values_dividend[][TEST_BIGNUM_WORDS] = {
        {0x8e020bca63c2d3b4, 0xf15d956d1119704c, 0x793bbdfa2cbe57d7, 0x51a13724b434b483, 0xda8f4665b027f674, 0xfab37c1f434754f2, 0x9352e2c1b6dc753e, 0x0675365166805884},
    };

    BN_ULONG test_values_divisor[][TEST_BIGNUM_WORDS] = {
        {0, 0, 0, 0, 0xffffffffffffffff, 0xffffffffffffffff, 0xffffffffffffffff, 0xfffffffefffffc2f},
    };

    reverse_order(test_values_dividend, test_values_divisor, sizeof(test_values_dividend) / (sizeof(BN_ULONG) * TEST_BIGNUM_WORDS));

    int sign_a[] = {0}; // Signs for 'a', add -1 for negative numbers as needed
    int sign_b[] = {0}; // Signs for 'b', add -1 for negative numbers as needed
    
    int num_tests = sizeof(test_values_dividend) / (sizeof(BN_ULONG) * TEST_BIGNUM_WORDS);

    int success = 0;

    for (int test = 0; test < num_tests; ++test) {
        BIGNUM dividend, divisor, quotient, remainder;
        init_zero(&dividend, TEST_BIGNUM_WORDS);
        init_zero(&divisor, TEST_BIGNUM_WORDS);
        init_zero(&quotient, TEST_BIGNUM_WORDS);
        init_zero(&remainder, TEST_BIGNUM_WORDS);

        // Initialize 'dividend' and 'divisor' with the test values
        for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            dividend.d[j] = test_values_dividend[test][j];
            divisor.d[j] = test_values_divisor[test][j];
        }
        dividend.top = find_top(&dividend, TEST_BIGNUM_WORDS);
        divisor.top = find_top(&divisor, TEST_BIGNUM_WORDS);

        dividend.neg = sign_a[test];
        divisor.neg = sign_b[test];

        printf("\n]==>> Test %d:\n", test);
        bn_print("dividend: ", &dividend);
        bn_print("n: ", &divisor);

        // Test division
        success = bn_div(&quotient, &remainder, &dividend, &divisor);
        // Print results
        if (success) {
            printf("Success\n");
        } else {
            printf("Failure\n");
        }
        bn_print("# quotient : ", &quotient);
        bn_print("# remainder: ", &remainder);
        printf("\n");
        // dividend
        // -------- = quotient, remainder
        // divisor
        // Multiplication back: quotient * divisor + remainder = dividend
        BIGNUM temp, product;
        init_zero(&temp, TEST_BIGNUM_WORDS);
        init_zero(&product, TEST_BIGNUM_WORDS);
        bn_mul(&quotient, &divisor, &product);
        // print product
        bn_print("product: ", &product);
        // add remainder
        bn_add(&temp, &product, &remainder);
        // print temp
        bn_print("temp: ", &temp);
        // print dividend
        bn_print("initial dividend: ", &dividend);
    }
      
}

// Main function
int main() {
    testKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipDeviceSynchronize();
    return 0;
}