#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

#define TEST_BIGNUM_WORDS 8

__device__ void reverse_order(BN_ULONG test_values_a[][TEST_BIGNUM_WORDS], BN_ULONG test_values_b[][TEST_BIGNUM_WORDS], size_t num_rows) {
    for (size_t i = 0; i < num_rows; i++) {
        for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
            BN_ULONG temp_a = test_values_a[i][j];
            test_values_a[i][j] = test_values_a[i][TEST_BIGNUM_WORDS - 1 - j];
            test_values_a[i][TEST_BIGNUM_WORDS - 1 - j] = temp_a;

            BN_ULONG temp_b = test_values_b[i][j];
            test_values_b[i][j] = test_values_b[i][TEST_BIGNUM_WORDS - 1 - j];
            test_values_b[i][TEST_BIGNUM_WORDS - 1 - j] = temp_b;
        }
    }
}

__global__ void testKernel() {
    printf("++ testKernel for bn_mod ++\n");
    BN_ULONG test_values_a[][TEST_BIGNUM_WORDS] = {
        {0xa9d76a4234a8ded, 0x7af964ec3f6f871b, 0xe09d7f67cc580732, 0x3b11b98c6222abbb, 0x0bdfd291448c33e6, 0xa46834fe88684cf0, 0x5106877163ee71eb, 0x5186b6de04720283},
        {0, 0, 0, 0x1, 0x2E09165B257A4C3E, 0x52C9F4FAA6322C65, 0x898D5D622CB3EEFF, 0x55DA7F062F1B85C0},
    };

    BN_ULONG test_values_n[][TEST_BIGNUM_WORDS] = {
        {0, 0, 0, 0, 0xffffffffffffffff, 0xffffffffffffffff, 0xffffffffffffffff, 0xfffffffefffffc2f},
        {0, 0, 0, 0, 0xFFFFFFFFFFFFFFFF, 0xFFFFFFFFFFFFFFFE, 0xBAAEDCE6AF48A03B, 0xBFD25E8CD0364141},
    };

    int mod;

    // 0 for positive, 1 for negative
    int sign_a[] = {0,0};
    int sign_n[] = {0,0};
    
    reverse_order(test_values_a, test_values_n, sizeof(test_values_a) / (sizeof(BN_ULONG) * TEST_BIGNUM_WORDS));
    
    int num_tests = sizeof(test_values_a) / (sizeof(BN_ULONG) * TEST_BIGNUM_WORDS);
    for (int test = 0; test < num_tests; ++test) {
        BIGNUM value_a, value_n, remainder;
        init_zero(&value_a, TEST_BIGNUM_WORDS);
        init_zero(&value_n, TEST_BIGNUM_WORDS);
        init_zero(&remainder, TEST_BIGNUM_WORDS);

        // Initialize 'value_a' and 'value_n' with the test values
        for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            value_a.d[j] = test_values_a[test][j];
            value_n.d[j] = test_values_n[test][j];
        }
        value_a.top = find_top(&value_a, TEST_BIGNUM_WORDS);
        value_n.top = find_top(&value_n, TEST_BIGNUM_WORDS);

        value_a.neg = sign_a[test];
        value_n.neg = sign_n[test];

        printf("\n]================>> Test %d:\n", test);
        bn_print("a: ", &value_a);
        bn_print("n: ", &value_n);

        mod = bn_mod(&remainder, &value_a, &value_n);

        // Print results
        bn_print("remainder: ", &remainder);
        bn_print("a : ", &value_a);
        bn_print("n : ", &value_n);
        printf("mod: %d\n", mod);
    }
}

// Main function
int main() {
    testKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipDeviceSynchronize();
    return 0;
}