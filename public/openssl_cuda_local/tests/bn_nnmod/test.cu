#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

#define TEST_BIGNUM_WORDS 10

__device__ void reverse_order(BN_ULONG test_values_a[][TEST_BIGNUM_WORDS], BN_ULONG test_values_b[][TEST_BIGNUM_WORDS], size_t num_rows) {
    for (size_t i = 0; i < num_rows; i++) {
        for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
            BN_ULONG temp_a = test_values_a[i][j];
            test_values_a[i][j] = test_values_a[i][TEST_BIGNUM_WORDS - 1 - j];
            test_values_a[i][TEST_BIGNUM_WORDS - 1 - j] = temp_a;

            BN_ULONG temp_b = test_values_b[i][j];
            test_values_b[i][j] = test_values_b[i][TEST_BIGNUM_WORDS - 1 - j];
            test_values_b[i][TEST_BIGNUM_WORDS - 1 - j] = temp_b;
        }
    }
}

__global__ void testKernel() {
    printf("++ testKernel for bn_mod ++\n");
    // c17747b1566d9fe8ab7087e3f0c50175b788a1c84f4c756c405000a0ca2248e1
    BN_ULONG test_values_a[][MAX_BIGNUM_SIZE] = {
        {0,0,0,0,0,0,0xc17747b1566d9fe8, 0xab7087e3f0c50175, 0xb788a1c84f4c756c, 0x405000a0ca2248e1}
        
    };
    // 6c91cea9cf0cac55a7596d16b56d2aefd204bb99dd677993158a7e6564f93cdf
    BN_ULONG test_values_n[][MAX_BIGNUM_SIZE] = {
        {0,0,0,0,0,0,0x6c91cea9cf0cac55, 0xa7596d16b56d2aef, 0xd204bb99dd677993, 0x158a7e6564f93cdf}
    };

    int mod;

    // 0 for positive, 1 for negative
    int sign_a[] = {0};
    int sign_n[] = {0};
    
    reverse_order(test_values_a, test_values_n, sizeof(test_values_a) / (sizeof(BN_ULONG) * TEST_BIGNUM_WORDS));
    
    int num_tests = sizeof(test_values_a) / (sizeof(BN_ULONG) * TEST_BIGNUM_WORDS);
    for (int test = 0; test < num_tests; ++test) {
        BIGNUM value_a, value_n, remainder;
        init_zero(&value_a);
        init_zero(&value_n);
        init_zero(&remainder);

        // Initialize 'value_a' and 'value_n' with the test values
        for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            value_a.d[j] = test_values_a[test][j];
            value_n.d[j] = test_values_n[test][j];
        }
        value_a.top = find_top(&value_a);
        value_n.top = find_top(&value_n);

        value_a.neg = sign_a[test];
        value_n.neg = sign_n[test];

        printf("\n]================>> Test %d:\n", test);
        bn_print("a: ", &value_a);
        bn_print("n: ", &value_n);

        mod = bn_mod(&remainder, &value_a, &value_n);

        // Print results
        bn_print("remainder: ", &remainder);
        printf("mod: %d\n", mod);
    }
}

// Main function
int main() {
    testKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipDeviceSynchronize();
    return 0;
}