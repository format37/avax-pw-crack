#include "hip/hip_runtime.h"
#include <fstream>
#include <iomanip>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

#define TEST_BIGNUM_WORDS 4

__device__ void reverse_order(BIGNUM *test_values_a) {
    for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
        BN_ULONG temp_a = test_values_a->d[j];
        test_values_a->d[j] = test_values_a->d[TEST_BIGNUM_WORDS - 1 - j];
        test_values_a->d[TEST_BIGNUM_WORDS - 1 - j] = temp_a;
    }
}

__device__ EC_POINT ec_point_scalar_mul(
    EC_POINT *point, 
    BIGNUM *scalar, 
    BIGNUM *curve_prime, 
    BIGNUM *curve_a
    ) {
    // debug_printf("++ ec_point_scalar_mul ++\n");
    // Print point
    bn_print(">> point x: ", &point->x);
    bn_print(">> point y: ", &point->y);
    bn_print(">> scalar: ", scalar);
    bn_print(">> curve_prime: ", curve_prime);
    bn_print(">> curve_a: ", curve_a);    
    
    EC_POINT current = *point; // This initializes the current point with the input point
    EC_POINT result; // Initialize the result variable, which accumulates the result
    EC_POINT tmp_result;
    EC_POINT tmp_a;
    EC_POINT tmp_b;                                     
    
    init_point_at_infinity(&result);                 // Initialize it to the point at infinity
    init_point_at_infinity(&tmp_result);                 // Initialize it to the point at infinity
    init_point_at_infinity(&tmp_a);                 // Initialize it to the point at infinity
    init_point_at_infinity(&tmp_b);                 // Initialize it to the point at infinity
    // printf("0: Interrupting for debug\n");
    
    // Convert scalar BIGNUM to an array of integers that's easy to iterate bit-wise
    unsigned int bits[256];                          // Assuming a 256-bit scalar
    bignum_to_bit_array(scalar, bits);               // You will need to implement bignum_to_bit_array()
    
    // printf("coef hex: %s\n", bignum_to_hex(scalar)); // Convert BIGNUM to hex string for printing
    bn_print("coef: ", scalar);  
    
    // int debug_counter = 1;    
    
    for (int i = 0; i < 256; i++) {                 // Assuming 256-bit scalars
        // printf("\n### Step: %d\n", i);
        // if (i<debug_counter) {
        //     // printf("0 x: %s\n", bignum_to_hex(&current.x));
        //     bn_print("0 current.x: ", &current.x);
        //     // printf("0 y: %s\n", bignum_to_hex(&current.y));
        //     bn_print("0 current.y: ", &current.y);
        // }
        

        if (bits[i]) {// If the i-th bit is set
            // printf("\n[0]\n");
            // printf("0: Interrupting for debug\n");
            // return result; // TODO: remove this
            // if (i<debug_counter) printf("# 0\n");
            // point_add(&result, &current, &result);  // Add current to the result
            // point_add(&result, &current, &result, &field_order);  // Add current to the result
            //point_add(&result, &current, &result, curve_order);  // Add current to the result

            // init tmp_result
            init_point_at_infinity(&tmp_result); 
            
            bn_print(">> point_add result.x: ", &result.x);
            bn_print(">> point_add result.y: ", &result.y);
            bn_print(">> point_add current.x: ", &current.x);
            bn_print(">> point_add current.y: ", &current.y);
            bn_print(">> curve_prime: ", curve_prime);
            bn_print(">> curve_a: ", curve_a);
            point_add(&tmp_result, &result, &current, curve_prime, curve_a);  // Add current to the result
            init_point_at_infinity(&result); // Reset result
            bn_copy(&result.x, &tmp_result.x);
            bn_copy(&result.y, &tmp_result.y);
            bn_print("<< point_add result.x: ", &result.x);
            bn_print("<< point_add result.y: ", &result.y);
            
            // if (i<debug_counter) printf("# b\n");
            // printf("1 x: %s\n", bignum_to_hex(&result.x));
            //  if (i<debug_counter) bn_print("1 result.x: ", &result.x);
            // printf("1 y: %s\n", bignum_to_hex(&result.y));
            //  if (i<debug_counter) bn_print("1 result.y: ", &result.y);
            // printf("\n");
            
        }
        // else {
        //     printf("1: Interrupting for debug\n");
        //     return result; // TODO: remove this
        // }
        // if (i<debug_counter) printf("# c\n");

        //point_double(&current, &current);           // Double current
        // point_double(&current, &current, &field_order);  // Double current and store the result in current
        // point_double(&current, &current, curve_order);

        // We don't need to double the point. We can just add it to itself.
        //point_add(&current, &current, &current, curve_order);
        // bn_print("\n>> [1] point_add current.x: ", &current.x);
        // bn_print(">> point_add current.y: ", &current.y);
        // bn_print(">> point_add result.x: ", &result.x);
        // bn_print(">> point_add result.y: ", &result.y);
        // bn_print(">> point_add curve_prime: ", curve_prime);
        // bn_print(">> point_add curve_a: ", curve_a);
        // printf("0: Interrupting for debug\n");
        // return result; // TODO: remove this
        // __device__ int point_add(
        //     EC_POINT *result, 
        //     EC_POINT *p1, 
        //     EC_POINT *p2, 
        //     BIGNUM *p, 
        //     BIGNUM *a
        // ) {
        // init tmp_result
        init_point_at_infinity(&tmp_result);
        // init tmp_a
        init_point_at_infinity(&tmp_a);
        // init tmp_b
        init_point_at_infinity(&tmp_b);
        // Copy current to tmp_a
        bn_copy(&tmp_a.x, &current.x);
        bn_copy(&tmp_a.y, &current.y);
        // Copy current to tmp_b
        bn_copy(&tmp_b.x, &current.x);
        bn_copy(&tmp_b.y, &current.y);

        // printf("\n[1]\n");
        bn_print(">> point_add tmp_a.x: ", &tmp_a.x);
        bn_print(">> point_add tmp_a.y: ", &tmp_a.y);
        bn_print(">> point_add tmp_b.x: ", &tmp_b.x);
        bn_print(">> point_add tmp_b.y: ", &tmp_b.y);
        bn_print(">> point_add tmp_result.x: ", &tmp_result.x);
        bn_print(">> point_add tmp_result.y: ", &tmp_result.y);
        // print curve_prime and curve_a
        bn_print(">> point_add curve_prime: ", curve_prime);
        bn_print(">> point_add curve_a: ", curve_a);

        

        point_add(&tmp_result, &tmp_a, &tmp_b, curve_prime, curve_a);  // Double current by adding to itself
        // ATTENTION: tmp_result is not related to result

        // printf("### Breaking at i: %d\n", i);
        // break; // TODO: remove this

        bn_print("\n<< point_add tmp_result.x (pp.x): ", &tmp_result.x);
        bn_print("<< point_add tmp_result.y (pp.y): ", &tmp_result.y);
        bn_print("<< point_add tmp_a.x (p1.x): ", &tmp_a.x);
        bn_print("<< point_add tmp_a.y (p1.y): ", &tmp_a.y);
        bn_print("<< point_add tmp_b.x (p2.x): ", &tmp_b.x);
        bn_print("<< point_add tmp_b.y (p2.y):", &tmp_b.y);
        bn_print("<< point_add curve_prime: ", curve_prime);
        bn_print("<< point_add curve_a: ", curve_a);

        // Copy tmp_result to current
        bn_copy(&current.x, &tmp_result.x);
        bn_copy(&current.y, &tmp_result.y);
        bn_print("\n<< point_add current.x: ", &current.x);
        bn_print("<< point_add current.y: ", &current.y);

        // printf("2 x: %s\n", bignum_to_hex(&current.x));
        // if (i<debug_counter) bn_print("2 current.x: ", &current.x);
        // printf("2 y: %s\n", bignum_to_hex(&current.y));
        // if (i<debug_counter) bn_print("2 current.y: ", &current.y);
        // if (i>1) {
        //     printf("### Breaking at i: %d\n", i);
        //     break; // TODO: remove this        
        // }
    }

    // // printf("Final x: %s\n", bignum_to_hex(&result.x));
    // bn_print("Final x: ", &result.x);
    // // printf("Final y: %s\n", bignum_to_hex(&result.y));
    // bn_print("Final y: ", &result.y);
    // printf("-- ec_point_scalar_mul --\n");
    return result;
}
// Public key derivation --

__global__ void testKernel(BIGNUM* d_private_keys, EC_POINT* d_public_keys) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Global thread ID
    clock_t start = clock64();
    // printf("Thread %d - Starting execution\n", tid);

    // Addition
    BIGNUM a;
    BIGNUM b;
    BIGNUM curveOrder;
    BIGNUM newKey;

    init_zero(&a);
    init_zero(&b);
    init_zero(&curveOrder);
    init_zero(&newKey);

    // BN_ULONG a_d[4];
    // BN_ULONG b_d[4];

    // Initialize a
    // C17747B1566D9FE8AB7087E3F0C50175B788A1C84F4C756C405000A0CA2248E1
    BN_ULONG a_values[MAX_BIGNUM_SIZE] = {
        0xC17747B1566D9FE8,
        0xAB7087E3F0C50175,
        0xB788A1C84F4C756C,
        0x405000A0CA2248E1
        };
    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            a.d[j] = a_values[j];
        }
    a.neg = 0;
    a.top = 4;

    // Initialize b
    // 6C91CEA9CF0CAC55A7596D16B56D2AEFD204BB99DD677993158A7E6564F93CDF
    BN_ULONG b_values[MAX_BIGNUM_SIZE] = {
        0x6C91CEA9CF0CAC55,
        0xA7596D16B56D2AEF,
        0xD204BB99DD677993,
        0x158A7E6564F93CDF
        };
    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            b.d[j] = b_values[j];
        }
    b.neg = 0;
    b.top = 4;

    BN_ULONG curveOrder_values[MAX_BIGNUM_SIZE] = {
        0xffffffffffffffff,
        0xFFFFFFFFFFFFFFFE,
        0xBAAEDCE6AF48A03B,
        0xBFD25E8CD0364141
        };

    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            curveOrder.d[j] = curveOrder_values[j];
        }

    // Initialize curveOrder_d
    // FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141
    reverse_order(&a);
    reverse_order(&b);
    reverse_order(&curveOrder);

    // Print inputs
    // bn_print(">> bn_add a: ", &a);
    // bn_print(">> bn_add b: ", &b);

    // Add A and B
    //bn_print(">> bn_add newKey: ", &newKey);
    bn_add(&newKey, &a, &b); // result = a + b
    // bn_print("<< bn_add newKey: ", &newKey);

    // Modular Reduction
    BIGNUM m;
    init_zero(&m);
    m.d[0] = 0x64; // 100
    
    BIGNUM tmp;
    init_zero(&tmp);
    bn_copy(&tmp, &newKey);
    // bn_print("\n>> bn_mod tmp: ", &tmp);
    // bn_print(">> curveOrder: ", &curveOrder);
    bn_mod(&newKey, &tmp, &curveOrder); // a = b mod c
    // bn_print("<< bn_mod newKey: ", &newKey);
    // printf("(expected): 2E09165B257A4C3E52C9F4FAA6322C66CEDE807B7D6B4EC3960820795EE5447F\n");
    // bn_print("\nPrivate key: ", &newKey);
    // printf("Thread %d - After initialization\n", tid);
    // bn_print_constant("Private key: ", &newKey, tid);
    // Derive the public key
    // printf("\nDeriving the public key..\n");
    // Initialize constants
    init_zero(&CURVE_A);
    
    // For secp256k1, CURVE_B should be initialized to 7 rather than 0
    init_zero(&CURVE_B);
    CURVE_B.d[0] = 0x7;

    BN_ULONG CURVE_GX_values[MAX_BIGNUM_SIZE] = {
        0x79BE667EF9DCBBAC,
        0x55A06295CE870B07,
        0x029BFCDB2DCE28D9,
        0x59F2815B16F81798
        };
    for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
            CURVE_GX_d[j] = CURVE_GX_values[j];
        }

    // Generator y coordinate
    // BIGNUM CURVE_GY;
    BN_ULONG CURVE_GY_values[MAX_BIGNUM_SIZE] = {
        0x483ADA7726A3C465,
        0x5DA4FBFC0E1108A8,
        0xFD17B448A6855419,
        0x9C47D08FFB10D4B8
        };
    for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
            CURVE_GY_d[j] = CURVE_GY_values[j];
        }

    // Initialize generator
    EC_POINT G;
    init_zero(&G.x);
    init_zero(&G.y);
    for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
            G.x.d[j] = CURVE_GX_values[j];
            G.y.d[j] = CURVE_GY_values[j];
        }
    // reverse
    reverse_order(&G.x);
    reverse_order(&G.y);
    // find top
    G.x.top = find_top(&G.x);
    G.y.top = find_top(&G.y);

    init_zero(&CURVE_P);
    //bn_copy(&CURVE_P, &curveOrder); // CURVE_P is curveOrder_d
    // Init curve prime
    // fffffffffffffffffffffffffffffffffffffffffffffffffffffffefffffc2f
    BN_ULONG CURVE_P_values[MAX_BIGNUM_SIZE] = {
        0xFFFFFFFFFFFFFFFF,
        0xFFFFFFFFFFFFFFFF,
        0xFFFFFFFFFFFFFFFF,
        0xFFFFFFFEFFFFFC2F,
        0,0,0,0        
        };
    for (int j = 0; j < MAX_BIGNUM_SIZE; ++j) {
            CURVE_P.d[j] = CURVE_P_values[j];
        }
    // reverse
    reverse_order(&CURVE_P);
    // find top
    CURVE_P.top = find_top(&CURVE_P);
    
    // Derive public key 
    EC_POINT publicKey = ec_point_scalar_mul(&G, &newKey, &CURVE_P, &CURVE_A);

    // Store the results in global memory for test purposes
    bn_copy(&d_private_keys[tid], &newKey);
    bn_copy(&d_public_keys[tid].x, &publicKey.x);
    bn_copy(&d_public_keys[tid].y, &publicKey.y);
    
    // Print public key
    // printf("Thread %d - After public key derivation\n", tid);
    // bn_print_constant("Public key x: ", &publicKey.x, tid);
    // bn_print_constant("Public key y: ", &publicKey.y, tid);
    record_function(FN_MAIN, start);
    // Only print performance report for thread 0 to avoid clutter
    if (tid == 0) {
        print_performance_report();
    }
}

int main() {
    
    const int THREADS_PER_BLOCK = 2;
    // const int THREADS_PER_BLOCK = 192; // 344 seconds
    // const int THREADS_PER_BLOCK = 200; // stuck
    // const int THREADS_PER_BLOCK = 224; // stuck
    // const int THREADS_PER_BLOCK = 256; // A good balance between occupancy and flexibility
    
    const int NUM_BLOCKS = 128; // One block per SM
    
    const int TOTAL_THREADS = THREADS_PER_BLOCK * NUM_BLOCKS; // 32,768 total threads
    // Allocate memory for results
    BIGNUM* h_private_keys = new BIGNUM[TOTAL_THREADS];
    EC_POINT* h_public_keys = new EC_POINT[TOTAL_THREADS];
    
    BIGNUM* d_private_keys;
    EC_POINT* d_public_keys;

    // Allocate device memory
    hipMalloc(&d_private_keys, TOTAL_THREADS * sizeof(BIGNUM));
    hipMalloc(&d_public_keys, TOTAL_THREADS * sizeof(EC_POINT));

    // Launch kernel
    testKernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_private_keys, d_public_keys);

    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Copy results back to host
    hipMemcpy(h_private_keys, d_private_keys, TOTAL_THREADS * sizeof(BIGNUM), hipMemcpyDeviceToHost);
    hipMemcpy(h_public_keys, d_public_keys, TOTAL_THREADS * sizeof(EC_POINT), hipMemcpyDeviceToHost);

    // Save results to CSV file
    std::ofstream outfile("all_results.csv");
    outfile << "Thread,Key,Value\n";

    for (int i = 0; i < TOTAL_THREADS; i++) {
        outfile << i << ",Private Key,";
        for (int j = MAX_BIGNUM_SIZE - 1; j >= 0; j--) {
            outfile << std::setfill('0') << std::setw(16) << std::hex << h_private_keys[i].d[j];
        }
        outfile << "\n";

        outfile << i << ",Public Key X,";
        for (int j = MAX_BIGNUM_SIZE - 1; j >= 0; j--) {
            outfile << std::setfill('0') << std::setw(16) << std::hex << h_public_keys[i].x.d[j];
        }
        outfile << "\n";

        outfile << i << ",Public Key Y,";
        for (int j = MAX_BIGNUM_SIZE - 1; j >= 0; j--) {
            outfile << std::setfill('0') << std::setw(16) << std::hex << h_public_keys[i].y.d[j];
        }
        outfile << "\n";
    }

    outfile.close();

    // Free memory
    delete[] h_private_keys;
    delete[] h_public_keys;
    hipFree(d_private_keys);
    hipFree(d_public_keys);

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}