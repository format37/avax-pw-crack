#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

#define TEST_BIGNUM_WORDS 4

__device__ void reverse_order(BIGNUM *test_values_a) {
    for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
        BN_ULONG temp_a = test_values_a->d[j];
        test_values_a->d[j] = test_values_a->d[TEST_BIGNUM_WORDS - 1 - j];
        test_values_a->d[TEST_BIGNUM_WORDS - 1 - j] = temp_a;
    }
}

//__device__ EC_POINT ec_point_scalar_mul(EC_POINT *point, BIGNUM *scalar, BIGNUM *curve_order) {
__device__ EC_POINT ec_point_scalar_mul(
    EC_POINT *point, 
    BIGNUM *scalar, 
    BIGNUM *curve_prime, 
    BIGNUM *curve_a
    ) {
    printf("++ ec_point_scalar_mul ++\n");
    // Print point
    bn_print(">> point x: ", &point->x);
    bn_print(">> point y: ", &point->y);
    bn_print(">> scalar: ", scalar);
    bn_print(">> curve_prime: ", curve_prime);
    bn_print(">> curve_a: ", curve_a);    

    EC_POINT current = *point;                       // This initializes the current point with the input point
    EC_POINT result;                                 // Initialize the result variable, which accumulates the result

    init_point_at_infinity(&result);                 // Initialize it to the point at infinity

    // Convert scalar BIGNUM to an array of integers that's easy to iterate bit-wise
    unsigned int bits[256];                          // Assuming a 256-bit scalar
    bignum_to_bit_array(scalar, bits);               // You will need to implement bignum_to_bit_array()
    
    // debug_printf("coef hex: %s\n", bignum_to_hex(scalar)); // Convert BIGNUM to hex string for printing
    bn_print("coef: ", scalar);  
    
    int debug_counter = 1;
    
    for (int i = 0; i < 256; i++) {                 // Assuming 256-bit scalars
        if (i<debug_counter) {
            // debug_printf("0 x: %s\n", bignum_to_hex(&current.x));
            bn_print("0 current.x: ", &current.x);
            // debug_printf("0 y: %s\n", bignum_to_hex(&current.y));
            bn_print("0 current.y: ", &current.y);
        }
        

        if (bits[i]) {// If the i-th bit is set
            printf("0: Interrupting for debug\n");
            return result; // TODO: remove this
            // if (i<debug_counter) printf("# 0\n");
            // point_add(&result, &current, &result);  // Add current to the result
            // point_add(&result, &current, &result, &field_order);  // Add current to the result
            //point_add(&result, &current, &result, curve_order);  // Add current to the result
            

            point_add(&result, &current, &result, curve_prime, curve_a);  // Add current to the result
            
            

             // if (i<debug_counter) printf("# b\n");
            // debug_printf("1 x: %s\n", bignum_to_hex(&result.x));
             if (i<debug_counter) bn_print("1 result.x: ", &result.x);
            // debug_printf("1 y: %s\n", bignum_to_hex(&result.y));
             if (i<debug_counter) bn_print("1 result.y: ", &result.y);

        }
        else {
            printf("1: Interrupting for debug\n");
            return result; // TODO: remove this
        }
        if (i<debug_counter) debug_printf("# c\n");

        //point_double(&current, &current);           // Double current
        // point_double(&current, &current, &field_order);  // Double current and store the result in current
        // point_double(&current, &current, curve_order);

        // We don't need to double the point. We can just add it to itself.
        //point_add(&current, &current, &current, curve_order);
        point_add(&current, &current, &current, curve_prime, curve_a);  // Double current by adding to itself

        // debug_printf("2 x: %s\n", bignum_to_hex(&current.x));
        if (i<debug_counter) bn_print("2 current.x: ", &current.x);
        // debug_printf("2 y: %s\n", bignum_to_hex(&current.y));
        if (i<debug_counter) bn_print("2 current.y: ", &current.y);
        //printf("BREAKING\n");
        // break; // TODO: remove this
    }

    // debug_printf("Final x: %s\n", bignum_to_hex(&result.x));
    bn_print("Final x: ", &result.x);
    // debug_printf("Final y: %s\n", bignum_to_hex(&result.y));
    bn_print("Final y: ", &result.y);
    printf("-- ec_point_scalar_mul --\n");
    return result;
}
// Public key derivation --

__global__ void testKernel() {

    // BN_CTX *ctx = BN_CTX_new();

    // return;

    // Addition
    BIGNUM a;
    BIGNUM b;
    BIGNUM curveOrder;
    BIGNUM newKey;

    init_zero(&a, MAX_BIGNUM_SIZE);
    init_zero(&b, MAX_BIGNUM_SIZE);
    init_zero(&curveOrder, MAX_BIGNUM_SIZE);
    init_zero(&newKey, MAX_BIGNUM_SIZE);

    BN_ULONG a_d[4];
    BN_ULONG b_d[4];
    //BN_ULONG curveOrder_d[4];

    // Initialize a
    // C17747B1566D9FE8AB7087E3F0C50175B788A1C84F4C756C405000A0CA2248E1
    a_d[0] = 0xC17747B1566D9FE8;
    a_d[1] = 0xAB7087E3F0C50175;
    a_d[2] = 0xB788A1C84F4C756C;
    a_d[3] = 0x405000A0CA2248E1; 
    a.d = a_d; 
    a.top = 4;
    a.neg = 0;

    // Initialize b
    // 6C91CEA9CF0CAC55A7596D16B56D2AEFD204BB99DD677993158A7E6564F93CDF
    b_d[0] = 0x6C91CEA9CF0CAC55;
    b_d[1] = 0xA7596D16B56D2AEF;
    b_d[2] = 0xD204BB99DD677993;
    b_d[3] = 0x158A7E6564F93CDF;
    b.d = b_d;
    b.top = 4;
    b.neg = 0;

    BN_ULONG curveOrder_values[MAX_BIGNUM_WORDS] = {
        0xffffffffffffffff,
        0xFFFFFFFFFFFFFFFE,
        0xBAAEDCE6AF48A03B,
        0xBFD25E8CD0364141
        };

    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            curveOrder.d[j] = curveOrder_values[j];
        }

    // Initialize curveOrder_d
    // FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141
    // curveOrder_d[0] = 0xFFFFFFFFFFFFFFFF;
    // curveOrder_d[1] = 0xFFFFFFFFFFFFFFFE;
    // curveOrder_d[2] = 0xBAAEDCE6AF48A03B;
    // curveOrder_d[3] = 0xBFD25E8CD0364141;
    // curveOrder.d = curveOrder_d;
    // curveOrder.neg = 0;
    // curveOrder.top = 4;

    reverse_order(&a);
    reverse_order(&b);
    reverse_order(&curveOrder);

    // Print inputs
    // bn_print(">> bn_add a: ", &a);
    // bn_print(">> bn_add b: ", &b);

    // Add A and B
    //bn_print(">> bn_add newKey: ", &newKey);
    bn_add(&newKey, &a, &b); // result = a + b
    // bn_print("<< bn_add newKey: ", &newKey);

    // Modular Reduction
    BIGNUM m;
    BN_ULONG m_d[4];
    for (int i = 0; i < 4; i++) m_d[i] = 0;
    m_d[0] = 0x64; // 100
    m.d = m_d;
    m.top = 1;
    m.neg = 0;
    
    BIGNUM tmp;
    init_zero(&tmp, MAX_BIGNUM_SIZE);
    bn_copy(&tmp, &newKey);
    // bn_print("\n>> bn_mod tmp: ", &tmp);
    // bn_print(">> curveOrder: ", &curveOrder);
    bn_mod(&newKey, &tmp, &curveOrder); // a = b mod c
    // bn_print("<< bn_mod newKey: ", &newKey);
    // printf("(expected): 2E09165B257A4C3E52C9F4FAA6322C66CEDE807B7D6B4EC3960820795EE5447F\n");
    bn_print("\Private key: ", &newKey);

    // Derive the public key
    printf("\nDeriving the public key..\n");
    // Initialize constants
    // CURVE_P is curveOrder_d
    // CURVE_P.d = curveOrder_d;
    // CURVE_P.top = 4;
    // CURVE_P.neg = 0;
    
    // for (int i = 0; i < 4; i++) CURVE_A_d[i] = 0;
    // CURVE_A.d = CURVE_A_d;
    // CURVE_A.top = 4;
    // CURVE_A.neg = 0;
    init_zero(&CURVE_A, MAX_BIGNUM_SIZE);
    
    // For secp256k1, CURVE_B should be initialized to 7 rather than 0
    for (int i = 0; i < 4; i++) CURVE_B_d[i] = 0;
    CURVE_B_d[0] = 0x7;
    CURVE_B.d = CURVE_B_d;
    CURVE_B.top = 4;
    CURVE_B.neg = 0;

    // Generator x coordinate
    CURVE_GX_d[0] = 0x79BE667EF9DCBBAC;
    CURVE_GX_d[1] = 0x55A06295CE870B07;
    CURVE_GX_d[2] = 0x029BFCDB2DCE28D9;
    CURVE_GX_d[3] = 0x59F2815B16F81798; 

    // Generator y coordinate
    BIGNUM CURVE_GY;
    BN_ULONG CURVE_GY_d[4];
    CURVE_GY_d[0] = 0x483ADA7726A3C465;
    CURVE_GY_d[1] = 0x5DA4FBFC0E1108A8;
    CURVE_GY_d[2] = 0xFD17B448A6855419;
    CURVE_GY_d[3] = 0x9C47D08FFB10D4B8;

    // Initialize generator
    EC_POINT G;
    G.x.d = CURVE_GX_d; 
    G.y.d = CURVE_GY_d;
    // Set tops, negs
    G.x.top = 4;
    G.y.top = 4;
    G.x.neg = 0;
    G.y.neg = 0;

    // Derive public key 
    // EC_POINT publicKey = ec_point_scalar_mul(&G, &newKey, &curveOrder);    
    
    bn_copy(&CURVE_P, &curveOrder);

    EC_POINT publicKey = ec_point_scalar_mul(&G, &newKey, &CURVE_P, &CURVE_A);
    
    
    // ec_point_scalar_mul / point_add / mod_mul / bn_mod <= Issue

    // Print public key
    printf("Public key:\n");
    bn_print("Public key x: ", &publicKey.x);
    bn_print("Public key y: ", &publicKey.y);


    // BN_CTX_free(ctx);

}

// Main function
int main() {
    testKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }
    hipDeviceSynchronize();
    return 0;
}