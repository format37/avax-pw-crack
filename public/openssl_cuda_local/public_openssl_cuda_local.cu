#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "bignum.h"

#define TEST_BIGNUM_WORDS 4

__device__ void reverse_order(BIGNUM *test_values_a) {
    for (size_t j = 0; j < TEST_BIGNUM_WORDS / 2; j++) {
        BN_ULONG temp_a = test_values_a->d[j];
        test_values_a->d[j] = test_values_a->d[TEST_BIGNUM_WORDS - 1 - j];
        test_values_a->d[TEST_BIGNUM_WORDS - 1 - j] = temp_a;
    }
}

//__device__ EC_POINT ec_point_scalar_mul(EC_POINT *point, BIGNUM *scalar, BIGNUM *curve_order) {
__device__ EC_POINT ec_point_scalar_mul(
    EC_POINT *point, 
    BIGNUM *scalar, 
    BIGNUM *curve_prime, 
    BIGNUM *curve_a
    ) {
    debug_printf("++ ec_point_scalar_mul ++\n");
    // Print point
    bn_print(">> point x: ", &point->x);
    bn_print(">> point y: ", &point->y);
    bn_print(">> scalar: ", scalar);
    bn_print(">> curve_prime: ", curve_prime);
    bn_print(">> curve_a: ", curve_a);    
    
    EC_POINT current = *point; // This initializes the current point with the input point
    EC_POINT result; // Initialize the result variable, which accumulates the result
    EC_POINT tmp_result;
    EC_POINT tmp_a;
    EC_POINT tmp_b;                                     
    
    init_point_at_infinity(&result);                 // Initialize it to the point at infinity
    init_point_at_infinity(&tmp_result);                 // Initialize it to the point at infinity
    init_point_at_infinity(&tmp_a);                 // Initialize it to the point at infinity
    init_point_at_infinity(&tmp_b);                 // Initialize it to the point at infinity
    // printf("0: Interrupting for debug\n");
    // return result; // TODO: remove this
    // Convert scalar BIGNUM to an array of integers that's easy to iterate bit-wise
    unsigned int bits[256];                          // Assuming a 256-bit scalar
    bignum_to_bit_array(scalar, bits);               // You will need to implement bignum_to_bit_array()
    
    // printf("coef hex: %s\n", bignum_to_hex(scalar)); // Convert BIGNUM to hex string for printing
    bn_print("coef: ", scalar);  
    
    int debug_counter = 1;    
    
    for (int i = 0; i < 256; i++) {                 // Assuming 256-bit scalars
        printf("\n### Step: %d\n", i);
        // if (i<debug_counter) {
        //     // printf("0 x: %s\n", bignum_to_hex(&current.x));
        //     bn_print("0 current.x: ", &current.x);
        //     // printf("0 y: %s\n", bignum_to_hex(&current.y));
        //     bn_print("0 current.y: ", &current.y);
        // }
        

        if (bits[i]) {// If the i-th bit is set
            printf("\n[0]\n");
            // printf("0: Interrupting for debug\n");
            // return result; // TODO: remove this
            // if (i<debug_counter) printf("# 0\n");
            // point_add(&result, &current, &result);  // Add current to the result
            // point_add(&result, &current, &result, &field_order);  // Add current to the result
            //point_add(&result, &current, &result, curve_order);  // Add current to the result

            // init tmp_result
            init_point_at_infinity(&tmp_result); 
            
            bn_print(">> point_add result.x: ", &result.x);
            bn_print(">> point_add result.y: ", &result.y);
            bn_print(">> point_add current.x: ", &current.x);
            bn_print(">> point_add current.y: ", &current.y);
            bn_print(">> curve_prime: ", curve_prime);
            bn_print(">> curve_a: ", curve_a);
            point_add(&tmp_result, &result, &current, curve_prime, curve_a);  // Add current to the result
            init_point_at_infinity(&result); // Reset result
            bn_copy(&result.x, &tmp_result.x);
            bn_copy(&result.y, &tmp_result.y);
            bn_print("<< point_add result.x: ", &result.x);
            bn_print("<< point_add result.y: ", &result.y);
            
            // if (i<debug_counter) printf("# b\n");
            // printf("1 x: %s\n", bignum_to_hex(&result.x));
            //  if (i<debug_counter) bn_print("1 result.x: ", &result.x);
            // printf("1 y: %s\n", bignum_to_hex(&result.y));
            //  if (i<debug_counter) bn_print("1 result.y: ", &result.y);
            printf("\n");
            
        }
        // else {
        //     printf("1: Interrupting for debug\n");
        //     return result; // TODO: remove this
        // }
        // if (i<debug_counter) printf("# c\n");

        //point_double(&current, &current);           // Double current
        // point_double(&current, &current, &field_order);  // Double current and store the result in current
        // point_double(&current, &current, curve_order);

        // We don't need to double the point. We can just add it to itself.
        //point_add(&current, &current, &current, curve_order);
        // bn_print("\n>> [1] point_add current.x: ", &current.x);
        // bn_print(">> point_add current.y: ", &current.y);
        // bn_print(">> point_add result.x: ", &result.x);
        // bn_print(">> point_add result.y: ", &result.y);
        // bn_print(">> point_add curve_prime: ", curve_prime);
        // bn_print(">> point_add curve_a: ", curve_a);
        // printf("0: Interrupting for debug\n");
        // return result; // TODO: remove this
        // __device__ int point_add(
        //     EC_POINT *result, 
        //     EC_POINT *p1, 
        //     EC_POINT *p2, 
        //     BIGNUM *p, 
        //     BIGNUM *a
        // ) {
        // init tmp_result
        init_point_at_infinity(&tmp_result);
        // init tmp_a
        init_point_at_infinity(&tmp_a);
        // init tmp_b
        init_point_at_infinity(&tmp_b);
        // Copy current to tmp_a
        bn_copy(&tmp_a.x, &current.x);
        bn_copy(&tmp_a.y, &current.y);
        // Copy current to tmp_b
        bn_copy(&tmp_b.x, &current.x);
        bn_copy(&tmp_b.y, &current.y);

        printf("\n[1]\n");
        bn_print(">> point_add tmp_a.x: ", &tmp_a.x);
        bn_print(">> point_add tmp_a.y: ", &tmp_a.y);
        bn_print(">> point_add tmp_b.x: ", &tmp_b.x);
        bn_print(">> point_add tmp_b.y: ", &tmp_b.y);
        bn_print(">> point_add tmp_result.x: ", &tmp_result.x);
        bn_print(">> point_add tmp_result.y: ", &tmp_result.y);
        // print curve_prime and curve_a
        bn_print(">> point_add curve_prime: ", curve_prime);
        bn_print(">> point_add curve_a: ", curve_a);

        

        point_add(&tmp_result, &tmp_a, &tmp_b, curve_prime, curve_a);  // Double current by adding to itself
        // ATTENTION: tmp_result is not related to result

        // printf("### Breaking at i: %d\n", i);
        // break; // TODO: remove this

        bn_print("\n<< point_add tmp_result.x (pp.x): ", &tmp_result.x);
        bn_print("<< point_add tmp_result.y (pp.y): ", &tmp_result.y);
        bn_print("<< point_add tmp_a.x (p1.x): ", &tmp_a.x);
        bn_print("<< point_add tmp_a.y (p1.y): ", &tmp_a.y);
        bn_print("<< point_add tmp_b.x (p2.x): ", &tmp_b.x);
        bn_print("<< point_add tmp_b.y (p2.y):", &tmp_b.y);
        bn_print("<< point_add curve_prime: ", curve_prime);
        bn_print("<< point_add curve_a: ", curve_a);

        // Copy tmp_result to current
        bn_copy(&current.x, &tmp_result.x);
        bn_copy(&current.y, &tmp_result.y);
        bn_print("\n<< point_add current.x: ", &current.x);
        bn_print("<< point_add current.y: ", &current.y);

        // printf("2 x: %s\n", bignum_to_hex(&current.x));
        // if (i<debug_counter) bn_print("2 current.x: ", &current.x);
        // printf("2 y: %s\n", bignum_to_hex(&current.y));
        // if (i<debug_counter) bn_print("2 current.y: ", &current.y);
        // if (i>1) {
        //     printf("### Breaking at i: %d\n", i);
        //     break; // TODO: remove this        
        // }
    }

    // printf("Final x: %s\n", bignum_to_hex(&result.x));
    bn_print("Final x: ", &result.x);
    // printf("Final y: %s\n", bignum_to_hex(&result.y));
    bn_print("Final y: ", &result.y);
    printf("-- ec_point_scalar_mul --\n");
    return result;
}
// Public key derivation --

__global__ void testKernel() {
    clock_t start = clock64();
    // BN_CTX *ctx = BN_CTX_new();

    // return;

    // Addition
    BIGNUM a;
    BIGNUM b;
    BIGNUM curveOrder;
    BIGNUM newKey;

    init_zero(&a, MAX_BIGNUM_SIZE);
    init_zero(&b, MAX_BIGNUM_SIZE);
    init_zero(&curveOrder, MAX_BIGNUM_SIZE);
    init_zero(&newKey, MAX_BIGNUM_SIZE);

    BN_ULONG a_d[4];
    BN_ULONG b_d[4];
    //BN_ULONG curveOrder_d[4];

    // Initialize a
    // C17747B1566D9FE8AB7087E3F0C50175B788A1C84F4C756C405000A0CA2248E1
    BN_ULONG a_values[MAX_BIGNUM_WORDS] = {
        0xC17747B1566D9FE8,
        0xAB7087E3F0C50175,
        0xB788A1C84F4C756C,
        0x405000A0CA2248E1
        };
    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            a.d[j] = a_values[j];
        }
    a.neg = 0;
    a.top = 4;

    // Initialize b
    // 6C91CEA9CF0CAC55A7596D16B56D2AEFD204BB99DD677993158A7E6564F93CDF
    BN_ULONG b_values[MAX_BIGNUM_WORDS] = {
        0x6C91CEA9CF0CAC55,
        0xA7596D16B56D2AEF,
        0xD204BB99DD677993,
        0x158A7E6564F93CDF
        };
    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            b.d[j] = b_values[j];
        }
    b.neg = 0;
    b.top = 4;

    BN_ULONG curveOrder_values[MAX_BIGNUM_WORDS] = {
        0xffffffffffffffff,
        0xFFFFFFFFFFFFFFFE,
        0xBAAEDCE6AF48A03B,
        0xBFD25E8CD0364141
        };

    for (int j = 0; j < TEST_BIGNUM_WORDS; ++j) {
            curveOrder.d[j] = curveOrder_values[j];
        }

    // Initialize curveOrder_d
    // FFFFFFFFFFFFFFFFFFFFFFFFFFFFFFFEBAAEDCE6AF48A03BBFD25E8CD0364141

    // Print inputs
    // bn_print(">> reverse_order a: ", &a);
    // bn_print(">> reverse_order b: ", &b);

    reverse_order(&a);
    reverse_order(&b);
    reverse_order(&curveOrder);

    // Print inputs
    // bn_print(">> bn_add a: ", &a);
    // bn_print(">> bn_add b: ", &b);

    // Add A and B
    //bn_print(">> bn_add newKey: ", &newKey);
    bn_add(&newKey, &a, &b); // result = a + b
    // bn_print("<< bn_add newKey: ", &newKey);

    // Modular Reduction
    BIGNUM m;
    init_zero(&m, MAX_BIGNUM_SIZE);
    // BN_ULONG m_d[4];
    // for (int i = 0; i < 4; i++) m_d[i] = 0;
    // m_d[0] = 0x64; // 100
    // m.d = m_d;
    m.d[0] = 0x64; // 100
    // m.top = 1;
    // m.neg = 0;
    
    BIGNUM tmp;
    init_zero(&tmp, MAX_BIGNUM_SIZE);
    bn_copy(&tmp, &newKey);
    // bn_print("\n>> bn_mod tmp: ", &tmp);
    // bn_print(">> curveOrder: ", &curveOrder);
    bn_mod(&newKey, &tmp, &curveOrder); // a = b mod c
    // bn_print("<< bn_mod newKey: ", &newKey);
    // printf("(expected): 2E09165B257A4C3E52C9F4FAA6322C66CEDE807B7D6B4EC3960820795EE5447F\n");
    bn_print("\nPrivate key: ", &newKey);
    //return; // TODO: Remove this
    // Derive the public key
    printf("\nDeriving the public key..\n");
    // Initialize constants
    init_zero(&CURVE_A, MAX_BIGNUM_SIZE);
    
    // For secp256k1, CURVE_B should be initialized to 7 rather than 0
    // for (int i = 0; i < 4; i++) CURVE_B_d[i] = 0;
    init_zero(&CURVE_B, MAX_BIGNUM_SIZE);
    CURVE_B.d[0] = 0x7;
    // CURVE_B_d[0] = 0x7;
    // CURVE_B.d = CURVE_B_d;
    // CURVE_B.top = 4;
    // CURVE_B.neg = 0;

    BN_ULONG CURVE_GX_values[MAX_BIGNUM_WORDS] = {
        0x79BE667EF9DCBBAC,
        0x55A06295CE870B07,
        0x029BFCDB2DCE28D9,
        0x59F2815B16F81798
        };
    for (int j = 0; j < MAX_BIGNUM_WORDS; ++j) {
            CURVE_GX_d[j] = CURVE_GX_values[j];
        }

    // Generator y coordinate
    BIGNUM CURVE_GY;
    BN_ULONG CURVE_GY_values[MAX_BIGNUM_WORDS] = {
        0x483ADA7726A3C465,
        0x5DA4FBFC0E1108A8,
        0xFD17B448A6855419,
        0x9C47D08FFB10D4B8
        };
    for (int j = 0; j < MAX_BIGNUM_WORDS; ++j) {
            CURVE_GY_d[j] = CURVE_GY_values[j];
        }

    // Initialize generator
    EC_POINT G;
    init_zero(&G.x, MAX_BIGNUM_WORDS);
    init_zero(&G.y, MAX_BIGNUM_WORDS);
    // G.x.d = CURVE_GX_d; 
    // G.y.d = CURVE_GY_d;
    for (int j = 0; j < MAX_BIGNUM_WORDS; ++j) {
            G.x.d[j] = CURVE_GX_values[j];
            G.y.d[j] = CURVE_GY_values[j];
        }
    // Set tops, negs
    // G.x.top = 4;
    // G.y.top = 4;

    // G.x.neg = 0;
    // G.y.neg = 0;
    // reverse
    reverse_order(&G.x);
    reverse_order(&G.y);
    // find top
    G.x.top = find_top(&G.x, MAX_BIGNUM_WORDS);
    G.y.top = find_top(&G.y, MAX_BIGNUM_WORDS);

    init_zero(&CURVE_P, MAX_BIGNUM_SIZE);
    //bn_copy(&CURVE_P, &curveOrder); // CURVE_P is curveOrder_d
    // Init curve prime
    // fffffffffffffffffffffffffffffffffffffffffffffffffffffffefffffc2f
    // BN_ULONG CURVE_P_d[4];
    BN_ULONG CURVE_P_values[MAX_BIGNUM_SIZE] = {
        0xFFFFFFFFFFFFFFFF,
        0xFFFFFFFFFFFFFFFF,
        0xFFFFFFFFFFFFFFFF,
        0xFFFFFFFEFFFFFC2F,
        0,0,0,0        
        };
    // for (int j = 0; j < MAX_BIGNUM_WORDS; ++j) {
    //         CURVE_P_d[j] = CURVE_P_values[j];
    //     }
    // CURVE_P.d = CURVE_P_values;
    for (int j = 0; j < MAX_BIGNUM_WORDS; ++j) {
            CURVE_P.d[j] = CURVE_P_values[j];
        }
    // CURVE_P.top = 4;
    // CURVE_P.neg = 0;
    // reverse
    reverse_order(&CURVE_P);
    // find top
    CURVE_P.top = find_top(&CURVE_P, MAX_BIGNUM_WORDS);
    
    // Derive public key 
    // EC_POINT publicKey = ec_point_scalar_mul(&G, &newKey, &curveOrder);    
    EC_POINT publicKey = ec_point_scalar_mul(&G, &newKey, &CURVE_P, &CURVE_A);

    // Print public key
    printf("Public key:\n");
    bn_print("Public key x: ", &publicKey.x);
    bn_print("Public key y: ", &publicKey.y);
    // stat_report("main", start);
    record_function(FN_MAIN, start);
    print_performance_report();
}

// Main function
int main() {
    // FunctionStats* h_functionStats;
    // int* h_functionCount;
    // FunctionStats* d_functionStats;
    // int* d_functionCount;

    // // Allocate host memory
    // h_functionStats = (FunctionStats*)malloc(MAX_FUNCTIONS * sizeof(FunctionStats));
    // h_functionCount = (int*)malloc(sizeof(int));

    // // Allocate device memory
    // hipMalloc(&d_functionStats, MAX_FUNCTIONS * sizeof(FunctionStats));
    // hipMalloc(&d_functionCount, sizeof(int));

    // Initialize device memory
    // init_stats<<<1, 1>>>(d_functionStats, d_functionCount);

    testKernel<<<1, 1>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Copy results back to host
    // hipMemcpy(h_functionStats, d_functionStats, MAX_FUNCTIONS * sizeof(FunctionStats), hipMemcpyDeviceToHost);
    // hipMemcpy(h_functionCount, d_functionCount, sizeof(int), hipMemcpyDeviceToHost);

    // Process results...

    // // Free memory
    // free(h_functionStats);
    // free(h_functionCount);
    // hipFree(d_functionStats);
    // hipFree(d_functionCount);

    hipDeviceSynchronize();
    hipDeviceReset(); // For cuda-memcheck leak-check option
    return 0;
}